#include "hip/hip_runtime.h"
#include "./config.cuh"
#include "./common/types.hpp"
#include "./common/cuda_computation.cuh"
#include "./common/cuda_common.cuh"
#include <math.h>

template<class REAL, int halo>
__global__ void 
#ifndef BOX
kernel2d_restrict
#else
kernel2d_restrict_box
#endif
        (REAL* input,int width_y, int width_x, REAL* output) 
{
  stencilParaT;
  int l_x = blockDim.x * blockIdx.x + threadIdx.x;  
  int l_y = blockDim.y * blockIdx.y + threadIdx.y;

#ifndef BOX
  int c = l_x + l_y * width_x;
  // int w[halo];
  // int e[halo];
  // int n[halo];
  // int s[halo];
  // // _Pragma("unroll") 
  // for(int hl=0; hl<halo; hl++)
  // {
  //   w[hl] = max(0,l_x-1-hl)+l_y * width_x;
  //   e[hl] = min(width_x-1,l_x+1+hl)+l_y * width_x;
  //   s[hl] = l_x+max(0,l_y-1-hl) * width_x;;
  //   n[hl] = l_x+min(width_y-1,l_y+1+hl) * width_x;
  // }
  REAL sum=0;
  // _Pragma("unroll") 
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
  for(int hl=0; hl<halo; hl++)
  {
    // sum+=south[hl]*input[s[hl]];
    sum+=south[hl]*input[l_x+max(0,l_y-1-hl) * width_x];
  }
  // _Pragma("unroll") 
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
  for(int hl=0; hl<halo; hl++)
  {
    // sum+=west[hl]*input[w[hl]];
    sum+=west[hl]*input[max(0,l_x-1-hl)+l_y * width_x];
  }
  sum+=center*input[c];
  // _Pragma("unroll") 
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
  for(int hl=0; hl<halo; hl++)
  {
    // sum+=east[hl]*input[e[hl]];
    sum+=east[hl]*input[min(width_x-1,l_x+1+hl)+l_y * width_x];
  }
  // _Pragma("unroll") 
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
  for(int hl=0; hl<halo; hl++)
  {
    // sum+=north[hl]*input[n[hl]];
    sum+=north[hl]*input[l_x+min(width_y-1,l_y+1+hl) * width_x];
  }
  output[c]=sum;
  return;
#else
  // int vertical[HALO*2+1];
  // int horizontal[HALO*2+1];
  // #pragma unroll
  // for(int hl_y=-HALO; hl_y<=HALO; hl_y++)
  // {
  //   vertical[hl_y+HALO]=min(max(l_y+hl_y,0),width_y-1)*width_x;
  // }
  // #pragma unroll
  // for(int hl_x=-HALO; hl_x<=HALO; hl_x++)
  // {
  //   horizontal[hl_x+HALO]=min(max(l_x+hl_x,0),width_x-1);
  // }
  REAL sum=0;
  // #pragma unroll
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
  for(int hl_y=-HALO; hl_y<=HALO; hl_y++)
  {
#ifndef NAIVENVCC
  #pragma unroll 1
#endif
    for(int hl_x=-HALO; hl_x<=HALO; hl_x++)
    {
      sum+=filter[hl_y+HALO][hl_x+HALO]*
      input[min(max(l_y+hl_y,0),width_y-1)*width_x  + min(max(l_x+hl_x,0),width_x-1)];
    }
  }
  output[l_y*width_x  + l_x]=sum;
  return;
#endif
}

// template __global__ void kernel2d_restrict<double,1>(double*,int,int,double*);
// template __global__ void kernel2d_restrict<float,1>(float*,int,int,float*);
#ifndef BOX
PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_REFERENCE,HALO);
#else
PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_REFERENCE_BOX,HALO);
#endif
// PERKS_INITIALIZE_ALL_TYPE_WITH_HALO(PERKS_DECLARE_INITIONIZATION_REFERENCE,2);