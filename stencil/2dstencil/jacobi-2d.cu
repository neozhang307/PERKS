#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <hip/hip_cooperative_groups.h>
#include "stdio.h"
#include "assert.h"
#include "config.cuh" 
#include "./common/jacobi_cuda.cuh"
#include "./common/types.hpp"
#include "./common/cuda_common.cuh"
#include "./common/cuda_computation.cuh"



#ifdef GEN
#include "./genconfig.cuh"
#endif

//#ifndef REAL
//#define REAL float
//#endif
//configuration
#if defined(NAIVE)||defined(BASELINE)||defined(BASELINE_CM)
  #define TRADITIONLAUNCH
#endif
#if defined(GEN)||defined(PERSISTENT)
  #define PERSISTENTLAUNCH
#endif
#if defined PERSISTENTLAUNCH||defined(BASELINE_CM)
  #define PERSISTENTTHREAD
#endif
#if defined(BASELINE)||defined(BASELINE_CM)||defined(GEN)||defined(MIX)||defined(PERSISTENT)
  #define USEMAXSM
#endif

#ifdef __PRINT__ 
  #define WARMUPRUN
#endif

#ifndef RUNS
  #define RUNS (1)
#endif
namespace cg = cooperative_groups;


#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 }                                                                 \
}


__global__ void printptx(int *result)
{
  // printf("code is run in %d\n",PERKS_ARCH);
  result[0]=PERKS_ARCH;
}
void host_printptx(int&result)
{
  int*d_r;
  hipMalloc((void**)&d_r, sizeof(int));
  printptx<<<1,1>>>(d_r);
  hipMemcpy(&result, d_r, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

#ifndef RTILE_Y
#define RTILE_Y (8)
#endif



template<class REAL>
// void jacobi_iterative(REAL * h_input, int width_y, int width_x, REAL * __var_0__, int iteration, bool async=false){
void jacobi_iterative(REAL * h_input, int width_y, int width_x, REAL * __var_0__, int bdimx, int blkpsm, int iteration, bool async, bool useSM){
// extern "C" void jacobi_iterative(REAL * h_input, int width_y, int width_x, REAL * __var_0__, int iteration){
/* Host allocation Begin */
  int ptx;
  host_printptx(ptx);
  #ifndef __PRINT__
    printf("code is run in %d\n",ptx);
  #endif
/*************************************/
  if(ptx<800&&async==true)
  {
    printf("error async not support\n");//lower ptw not support 
    return;
  }
//initialization
#if defined(PERSISTENT)

  #ifndef BOX
  auto execute_kernel = async?kernel_persistent_baseline_async<REAL,RTILE_Y,HALO>: kernel_persistent_baseline<REAL,RTILE_Y,HALO>;
  #else
  auto execute_kernel = async?kernel_persistent_baseline_box_async<REAL,RTILE_Y,HALO>:kernel_persistent_baseline_box<REAL,RTILE_Y,HALO>;
  #endif
#endif
#if defined(BASELINE_CM)||defined(BASELINE)
  #ifndef BOX
    auto execute_kernel = async?kernel_baseline_async<REAL,RTILE_Y,HALO>:kernel_baseline<REAL,RTILE_Y,HALO>;
  #else
    auto execute_kernel = async?kernel_baseline_box_async<REAL,RTILE_Y,HALO>:kernel_baseline_box<REAL,RTILE_Y,HALO>;
  #endif
#endif
#ifdef NAIVE
  #ifndef BOX
    auto execute_kernel = kernel2d_restrict<REAL,HALO>;
  #else
    auto execute_kernel = kernel2d_restrict_box<REAL,HALO>;
  #endif
#endif 
#ifdef GEN
  #ifndef BOX
  auto execute_kernel = async?
          (useSM?kernel_general_async<REAL,RTILE_Y,HALO,REG_FOLDER_Y,true>:
          kernel_general_async<REAL,RTILE_Y,HALO,REG_FOLDER_Y,false>)
          :
          (useSM?kernel_general<REAL,RTILE_Y,HALO,REG_FOLDER_Y,true>:
            kernel_general<REAL,RTILE_Y,HALO,REG_FOLDER_Y,false>);
  #else
  auto execute_kernel = async?
          (useSM?kernel_general_box_async<REAL,RTILE_Y,HALO,REG_FOLDER_Y,true>:
            kernel_general_box_async<REAL,RTILE_Y,HALO,REG_FOLDER_Y,false>)
            :
          (useSM?kernel_general_box<REAL,RTILE_Y,HALO,REG_FOLDER_Y,true>:
            kernel_general_box<REAL,RTILE_Y,HALO,REG_FOLDER_Y,false>)
          ;
  #endif
  //auto execute_kernel = kernel_general<REAL,RTILE_Y,HALO,REG_FOLDER_Y,UseSMCache>;
#endif
  int sm_count;
  hipDeviceGetAttribute ( &sm_count, hipDeviceAttributeMultiprocessorCount,0 );
  
  //initialization input and output space
  REAL * input;
  hipMalloc(&input,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipMemcpy(input,h_input,sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyHostToDevice);
  REAL * __var_1__;
  hipMalloc(&__var_1__,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  REAL * __var_2__;
  hipMalloc(&__var_2__,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");

  //initialize tmp space for halo region
#if defined(GEN) || defined(MIX)|| defined(PERSISTENT)
  REAL * L2_cache3;
  REAL * L2_cache4;
  size_t L2_utage_2 = sizeof(REAL)*(width_y)*2*(width_x/bdimx)*HALO;
#ifndef __PRINT__
  printf("l2 cache used is %ld KB : 4096 KB \n",L2_utage_2*2/1024);
#endif
  hipMalloc(&L2_cache3,L2_utage_2*2);
  L2_cache4=L2_cache3+(width_y)*2*(width_x/bdimx)*HALO;
#endif

  //initialize shared memory
  int maxSharedMemory;
  hipDeviceGetAttribute (&maxSharedMemory, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,0 );
  //could not use all share memory in a100. so set it in default.
  int SharedMemoryUsed = maxSharedMemory-1024;

#if defined(USEMAXSM)
  hipFuncSetAttribute(reinterpret_cast<const void*>(execute_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, SharedMemoryUsed);
#endif

size_t executeSM = 0;
#ifndef NAIVE
  //shared memory used for compuation
  int basic_sm_space=(RTILE_Y+2*HALO)*(bdimx+2*HALO);
  size_t sharememory_basic=(1+basic_sm_space)*sizeof(REAL);
  executeSM = sharememory_basic;

#endif




#ifdef PERSISTENTTHREAD
  int numBlocksPerSm_current=0;

  hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, execute_kernel, bdimx, executeSM);
  if(blkpsm!=0)
  {
    numBlocksPerSm_current=min(numBlocksPerSm_current,blkpsm);
  }
  dim3 block_dim(bdimx);
  dim3 grid_dim(width_x/bdimx,sm_count*numBlocksPerSm_current/(width_x/bdimx));
  
  dim3 executeBlockDim=block_dim;
  dim3 executeGridDim=grid_dim;

  #ifndef __PRINT__
    printf("blk per sm is %d/%d\n", numBlocksPerSm_current,blkpsm);
    printf("grid is (%d,%d)\n", grid_dim.x, grid_dim.y);
  #endif

#endif 

  #ifdef PERSISTENT
    size_t max_sm_flder=0;
  #endif

  #define halo HALO
  #if defined(GEN) 

  size_t max_sm_flder=0;
  if(useSM)
  {
    max_sm_flder=(SharedMemoryUsed/sizeof(REAL)/numBlocksPerSm_current
                            -2*HALO*isBOX
                            -basic_sm_space
                            -2*HALO*(REG_FOLDER_Y)*RTILE_Y
                            -2*HALO*(bdimx+2*HALO))/(bdimx+4*HALO)/RTILE_Y;

    // size_t sm_cache_size = TOTAL_SM_CACHE_SPACE*sizeof(REAL);
    size_t sm_cache_size = (max_sm_flder*RTILE_Y+2*HALO)*(bdimx+2*HALO)*sizeof(REAL);
    size_t y_axle_halo = (HALO*2*((max_sm_flder + REG_FOLDER_Y)*RTILE_Y+isBOX))*sizeof(REAL);
    executeSM=sharememory_basic+y_axle_halo;
    executeSM+=sm_cache_size;
  }
  #undef halo

  #ifndef __PRINT__
    printf("the max flder is %ld and the total sm size is %ld/block\n", max_sm_flder, executeSM);
  #endif

  #endif


#ifdef NAIVE
  dim3 block_dim_1(MIN(width_x,bdimx),1);
  dim3 grid_dim_1(width_x/MIN(width_x,bdimx),width_y/1);

  dim3 executeBlockDim=block_dim_1;
  dim3 executeGridDim=grid_dim_1;
#endif
#ifdef BASELINE
  dim3 block_dim2(bdimx);
  dim3 grid_dim2(width_x/bdimx,MIN((sm_count*8*1024/bdimx)/(width_x/bdimx),width_y/RTILE_Y));
//  printf("<%d,%d,%d>",); 
  dim3 executeBlockDim=block_dim2;
  dim3 executeGridDim=grid_dim2;

#endif
//in order to get a better performance, warmup run is necessary.


#if defined(GEN) || defined(PERSISTENT)
  int l_iteration=iteration;
  void* ExecuteKernelArgs[] ={(void**)&input,(void**)&width_y,
    (void*)&width_x,(void*)&__var_2__,(void*)&L2_cache3,(void*)&L2_cache4,
    (void*)&l_iteration, (void*)&max_sm_flder};
  #ifdef WARMUPRUN
    int warmupiteration=1;
    void* KernelArgs_NULL[] ={(void**)&__var_2__,(void**)&width_y,
      (void*)&width_x,(void*)&__var_1__,(void*)&L2_cache3,(void*)&L2_cache4,
      (void*)&warmupiteration, (void *)&max_sm_flder};
  #endif
#endif

//@LINGQI: l2 cache setting not show performance difference. 
#if defined(GEN) && defined(L2PER)
    // REAL l2perused;
    // size_t inner_window_size = 30*1024*1024;
    // hipLaunchAttributeValue stream_attribute;
    // stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(L2_cache3);                  // Global Memory data pointer
    // stream_attribute.accessPolicyWindow.num_bytes = min(inner_window_size,L2_utage_2*2);                                   // Number of bytes for persistence access
    // stream_attribute.accessPolicyWindow.hitRatio  = 1;                                             // Hint for cache hit ratio
    // stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;                  // Persistence Property
    // stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;  

    // hipStreamSetAttribute(0, hipLaunchAttributeAccessPolicyWindow, &stream_attribute); 
    // cudaCtxResetPersistingL2Cache();
    // hipStreamSynchronize(0);
#endif

#ifdef WARMUPRUN
      cudaCheckError();
  #ifdef TRADITIONLAUNCH
      execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (__var_2__, width_y, width_x,__var_1__);
  #endif 

  #ifdef PERSISTENTLAUNCH
      hipLaunchCooperativeKernel((void*)execute_kernel, executeGridDim, executeBlockDim, KernelArgs_NULL, executeSM,0);
  #endif

#endif 

#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif

#ifdef PERSISTENTLAUNCH
  for(int i=0; i<RUNS; i++)
  {
    hipLaunchCooperativeKernel((void*)execute_kernel, 
            executeGridDim, executeBlockDim, 
            ExecuteKernelArgs, 
            //KernelArgs4,
            executeSM,0);
  }
#endif
#ifdef TRADITIONLAUNCH
  for(int i=0; i<RUNS; i++)
  {
    execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (input, width_y, width_x, __var_2__);

    for(int i=1; i<iteration; i++)
    {
       execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (__var_2__, width_y, width_x , __var_1__);
      REAL* tmp = __var_2__;
      __var_2__=__var_1__;
      __var_1__= tmp;
    }
  }
#endif


#ifdef CHECK
  hipDeviceSynchronize();
  cudaCheckError();
#endif

#ifndef __PRINT__  
  printf("sm_count is %d\n", sm_count);
  printf("MAX shared memory is %f KB but only use %f KB\n", maxSharedMemory/1024.0,SharedMemoryUsed/1024.0);
  printf(" shared meomory size is %ld KB\n", executeSM/1024);

#endif


#ifdef __PRINT__
  printf("%d\t%d\t%d\t",ptx,sizeof(REAL)/4,(int)async);
  printf("%d\t%d\t%d\t",width_x,width_y,iteration);
  printf("%d\t<%d,%d>\t%d\t",executeBlockDim.x,
        executeGridDim.x,executeGridDim.y,
        (executeGridDim.x)*(executeGridDim.y)/sm_count);
  printf("%f\t",(double)sharememory_basic/1024);
#endif

#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  #ifdef __PRINT__
    printf("%f\t%f\n",elapsedTime,(REAL)iteration*(width_y-2*HALO)*(width_x-2*HALO)/ (elapsedTime/RUNS)/1000/1000);
  #else
    printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime/RUNS);
    printf("[FORMA] Speed(GCells/s) : %lf\n",(REAL)iteration*(width_y)*(width_x)/ elapsedTime/1000/1000*RUNS);
    printf("[FORMA] Speed(GFLOPS/s) : %lf\n", (REAL)17*iteration*(width_y)*(width_x)/ elapsedTime/1000/1000*RUNS);
    printf("[FORMA] bandwidth(GB/s) : %lf\n", (REAL)sizeof(REAL)*iteration*((width_y)*(width_x)+width_x*width_y)/ elapsedTime/1000/1000*RUNS);
    printf("[FORMA] width_x:width_y=%d:%d\n",(int)width_x, (int)width_y);
    printf("[FORMA] gdimx:gdimy=%d:%d\n",(int)executeGridDim.x, (int)executeGridDim.y);
    #if defined(GEN) || defined(MIX)
      printf("[FORMA] cached width_x:width_y=%d:%d\n",(int)bdimx*grid_dim.x, (int)(max_sm_flder+REG_FOLDER_Y)*RTILE_Y*grid_dim.y);
      printf("[FORMA] cached b:sf:rf=%d:%d:%d\n", (int)RTILE_Y, (int)max_sm_flder, (int)REG_FOLDER_Y);
    #endif
  #endif

  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif


//finalization
#ifdef CHECK
  // printf("check error here*\n");
  hipDeviceSynchronize();
  cudaCheckError();
#endif
  hipDeviceSynchronize();
  cudaCheckError();
#if defined(GEN) || defined(PERSISTENT)
  if(iteration%2==1)
    hipMemcpy(__var_0__,__var_2__, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
  else
    hipMemcpy(__var_0__,input, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
#else
  hipMemcpy(__var_0__,__var_2__, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);

#if defined(GEN) || defined(PERSISTENT)
  hipFree(L2_cache3);
#endif

}

PERKS_INITIALIZE_ALL_TYPE(PERKS_DECLARE_INITIONIZATION_ITERATIVE);


