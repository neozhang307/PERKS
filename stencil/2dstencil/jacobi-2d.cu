#include "hip/hip_runtime.h"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <hip/hip_cooperative_groups.h>
#include "stdio.h"
#include "assert.h"
#include "config.cuh" 
#include "./common/jacobi_cuda.cuh"
#include "./common/types.hpp"
#include "./common/cuda_common.cuh"
#include "./common/cuda_computation.cuh"

#ifdef SMASYNC
  #if PERKS_ARCH<800 
    #error "unsupport architecture"
  #endif
  #include <cooperative_groups/memcpy_async.h>
  #include <cuda_pipeline.h>
#endif

#ifdef GEN
#include "./genconfig.cuh"
#endif

//#ifndef REAL
//#define REAL float
//#endif
//configuration
#if defined(NAIVE)||defined(BASELINE)||defined(BASELINE_CM)
  #define TRADITIONLAUNCH
#endif
#if defined(GEN)||defined(MIX)||defined(PERSISTENT)
  #define PERSISTENTLAUNCH
#endif
#if defined PERSISTENTLAUNCH||defined(BASELINE_CM)
  #define PERSISTENTTHREAD
#endif
#if defined(BASELINE)||defined(BASELINE_CM)||defined(GEN)||defined(MIX)||defined(PERSISTENT)
  #define USEMAXSM
#endif

#ifdef __PRINT__ 
  #define WARMUPRUN
#endif


// #define FORMA_MAX(a,b) ( (a) > (b) ? (a) : (b) )
// #define MAX(a,b) FORMA_MAX(a,b)
// #define FORMA_MIN(a,b) ( (a) < (b) ? (a) : (b) )
// #define MIN(a,b) FORMA_MIN(a,b)
// #define FORMA_CEIL(a,b) ( (a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1 )


//#define SM_TILE_X (TILE_X+2*(HALO))
//#ifndef FORMA_MAX_BLOCKDIM_0
//#define FORMA_MAX_BLOCKDIM_0 1024
//#endif
//#ifndef FORMA_MAX_BLOCKDIM_1
//#define FORMA_MAX_BLOCKDIM_1 1024
//#endif
//#ifndef FORMA_MAX_BLOCKDIM_2
//#define FORMA_MAX_BLOCKDIM_2 1024
//#endif

namespace cg = cooperative_groups;

// void Check_CUDA_Error(const char* message);

//direction of x axle is the same as thread index
//basic tiling: tiling unit for single thread

/*********************ARGUMENTS for PERKS*******************************/
// Here "Folder" means how many times of "tiling unit" is stored in given memory structure
// Shared Memory folder of basic tiling
// #ifndef SM_FOLER_Y
// #define SM_FOLER_Y (2)
// #endif
// Register Files folder of basic tiling
// #ifndef REG_FOLDER_Y
// #define REG_FOLDER_Y (6)
// #endif
// Total 
// #define ISINITI (true)
// #define NOTINITIAL (false)
// #define SYNC (true)
// #define NOSYNC (false)

// //#undef TILE_Y
// // #define USESM

// #ifdef USESM
//   #define USESMSET (true)
// #else
//   #define USESMSET (false)
// #endif


#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 }                                                                 \
}

// #ifndef BOX
// #define stencilParaT \
//   const REAL west[6]={12.0/118,9.0/118,3.0/118,2.0/118,5.0/118,6.0/118};\
//   const REAL east[6]={12.0/118,9.0/118,3.0/118,3.0/118,4.0/118,6.0/118};\
//   const REAL north[6]={5.0/118,7.0/118,5.0/118,4.0/118,3.0/118,2.0/118};\
//   const REAL south[6]={5.0/118,7.0/118,5.0/118,1.0/118,6.0/118,2.0/118};\
//   const REAL center=15.0/118;
// ;
//   #define stencilParaList const REAL west[6],const REAL east[6],const REAL north[6],const REAL south[6],const REAL center
//   #define stencilParaInput  west,east,north,south,center
//   #define R_PTR r_ptr[INPUTREG_SIZE]
//   #define isBOX (0)
// #else
//   #if Halo==1
//   #define stencilParaT \
//   const REAL filter[3][3] = {\
//     {7.0/118, 5.0/118, 9.0/118},\
//     {12.0/118,15.0/118,12.0/118},\
//     {9.0/118, 5.0/118, 7.0/118}\
//   };
//   #endif
//   #if Halo==2
//   #define stencilParaT \
//   const REAL filter[5][5] = {\
//     {1.0/118, 2.0/118, 3.0/118, 4.0/118, 5.0/118},\
//     {7.0/118, 7.0/118, 5.0/118, 7.0/118, 6.0/118},\
//     {8.0/118,12.0/118,15.0/118,12.0/118,12.0/118},\
//     {9.0/118, 9.0/118, 5.0/118, 7.0/118, 15.0/118},\
//     {10.0/118, 11.0/118, 12.0/118, 13.0/118, 14.0/118}\
//   };
//   #endif

//   #define stencilParaList const REAL filter[halo*2+1][halo*2+1]
//   #define stencilParaInput  filter
//   #define R_PTR r_ptr[2*halo+1][INPUTREG_SIZE]
//   #define isBOX (halo)
// #endif




// #ifdef GEN
// template<class REAL, int LOCAL_TILE_Y=RTILE_Y, int halo=Halo, int reg_folder_y=REG_FOLDER_Y, bool UseSMCache=USESMSET>
// __global__ void kernel_general(REAL * __restrict__ input, int width_y, int width_x, 
//   REAL * __restrict__ __var_4__, 
//   REAL * __restrict__ l2_cache_o,REAL * __restrict__ l2_cache_i,
//   int iteration,
//   int max_sm_flder)
// {
//   if(!UseSMCache) max_sm_flder=0;
//   #define UseRegCache (reg_folder_y!=0)
//   #ifdef BOX
//     #define SM2REG sm2regs
//     #define REG2REG regs2regs
//   #else
//     #define SM2REG sm2reg
//     #define REG2REG reg2reg
//   #endif
//   stencilParaT;
//   //basic pointer
//   cg::grid_group gg = cg::this_grid();
//   //extern __shared__ REAL sm[];
//   extern __shared__ char sm[];


//   const int total_sm_tile_y = LOCAL_TILE_Y*max_sm_flder;//SM_FOLER_Y;//consider how to automatically compute it later
//   const int total_reg_tile_y = LOCAL_TILE_Y*reg_folder_y;
//   const int total_tile_y = total_sm_tile_y+total_reg_tile_y;
//   const int total_reg_tile_y_with_halo = total_reg_tile_y+2*halo;

//   const int sizeof_rspace = total_reg_tile_y_with_halo;
//   const int sizeof_rbuffer = LOCAL_TILE_Y+2*halo;

//   const int tile_x = blockDim.x;
//   const int tile_x_with_halo = tile_x + 2*halo;
//   const int tile_y_with_halo = LOCAL_TILE_Y+2*halo;
//   const int basic_sm_space=tile_x_with_halo*tile_y_with_halo;

//   const int boundary_line_size = total_tile_y+isBOX;
//   const int e_step = 0;
//   const int w_step = boundary_line_size*halo;

//   REAL* sm_rbuffer =(REAL*)sm+1;

//   REAL* boundary_buffer = sm_rbuffer + basic_sm_space;
//   REAL* sm_space = boundary_buffer+(2*halo*boundary_line_size);//BOX need add additional stuffs. 


//   //boundary space
//   //register buffer space
//   //seems use much space than necessary when no use register version. 
//   register REAL r_space[total_reg_tile_y_with_halo];
// #ifndef BOX
//   register REAL r_smbuffer[2*halo+LOCAL_TILE_Y];
// #else
//   register REAL r_smbuffer[2*halo+1][2*halo+LOCAL_TILE_Y];
// #endif

//   const int tid = threadIdx.x;
//   // int ps_x = Halo + tid;
//   const int ps_y = halo;
//   const int ps_x = halo;
//  // const int tile_x_with_halo = blockDim.x + 2*halo;

//   const int p_x = blockIdx.x * tile_x ;

//   int blocksize_y=(width_y/gridDim.y);
//   int y_quotient = width_y%gridDim.y;
  
//   const int p_y =  blockIdx.y * (blocksize_y) + (blockIdx.y<=y_quotient?blockIdx.y:y_quotient);
//   blocksize_y += (blockIdx.y<y_quotient?1:0);
//   const int p_y_cache = p_y + (blocksize_y-total_reg_tile_y-total_sm_tile_y);

//   //load data global to register
//   // #pragma unroll
//   if(UseRegCache)
//   {
//     global2reg<REAL,sizeof_rspace,total_reg_tile_y>(input, r_space,
//                                               p_y_cache, width_y,
//                                               p_x+tid, width_x,
//                                               halo);
//   }
//   // load data global to sm
//   if(UseSMCache)
//   {
//     global2sm<REAL,0>(input,sm_space,
//                                         total_sm_tile_y,
//                                         p_y_cache+total_reg_tile_y, width_y,
//                                         p_x, width_x,
//                                         ps_y, ps_x, tile_x_with_halo,
//                                         tid);
//   }
//   //load ew boundary
//   if(UseRegCache||UseSMCache)
//   {
//     for(int local_y=tid; local_y<boundary_line_size&&p_y_cache + local_y<width_y; local_y+=blockDim.x)
//     {
//       for(int l_x=0; l_x<halo; l_x++)
//       {
//         //east
//         int global_x = p_x + tile_x + l_x;
//         global_x = MIN(width_x-1,global_x);
//         boundary_buffer[e_step+local_y + l_x*boundary_line_size] = input[(p_y_cache + local_y) * width_x + global_x];
//         //west
//         global_x = p_x - halo + l_x;
//         global_x = MAX(0,global_x);
//         boundary_buffer[w_step+local_y + l_x*boundary_line_size] =  input[(p_y_cache + local_y) * width_x + global_x];
//       }
//     }
//     // sdfa
//   }
//   __syncthreads();
//   for(int iter=0; iter<iteration; iter++)
//   {
//     int local_x=tid;
//     //prefetch the boundary data
//     //north south
//     {
//       //register
//       if(UseRegCache||UseSMCache)
//       {  // #pragma unroll
//         _Pragma("unroll")
//         for(int l_y=0; l_y<halo; l_y++)
//         {
//           int global_y = (p_y_cache-halo+l_y);
//           global_y=MAX(0,global_y);
//           //south
//           // if(UseRegCache)
//           // {
//           //   r_space[l_y]=input[(global_y) * width_x + p_x + tid];
//           // }
//           // else
//           // {
//           //   sm_space[(ps_y - halo + l_y) * tile_x_with_halo + tid + ps_x]=input[(global_y) * width_x + p_x + tid];
//           // }
//           global_y=(p_y_cache+(total_sm_tile_y+total_reg_tile_y)+l_y);
//           global_y=MIN(global_y,width_y-1);
//           //north
//           if(UseSMCache)
//           {
//             //need to deal with boundary
//             sm_space[(ps_y +total_sm_tile_y + l_y) * tile_x_with_halo + tid + ps_x]=(input[(global_y) * width_x + p_x + tid]);
//           }
//           else
//           {
//             //need to deal with boundary
//             r_space[total_reg_tile_y+halo+l_y]=(input[(global_y) * width_x + p_x + tid]);
//           }
//           if(UseRegCache && UseSMCache)
//           {
//             //north of register
//             r_space[total_reg_tile_y+halo+l_y]=sm_space[(ps_y+l_y) * tile_x_with_halo + tid + ps_x];
//             //south of sm
//             // sm_space[(ps_y - halo+l_y) * tile_x_with_halo + tid + ps_x]=r_space[total_reg_tile_y+l_y];
//           }
//         }
//       }
//     }

//     //computation of general space 
//     global2sm<REAL,halo,ISINITI,SYNC>(input, sm_rbuffer, 
//                                             halo*2,
//                                             p_y-halo, width_y,
//                                             p_x, width_x,
//                                             ps_y-halo, ps_x, tile_x_with_halo,
//                                             tid);

//     SM2REG<REAL,sizeof_rbuffer, halo*2,isBOX>(sm_rbuffer, r_smbuffer, 
//                                                     0,
//                                                     ps_x, tid,
//                                                     tile_x_with_halo);

//     for(int global_y=p_y; global_y<p_y_cache; global_y+=LOCAL_TILE_Y)
//     {

//       global2sm<REAL,halo>(input, sm_rbuffer,
//                                           LOCAL_TILE_Y, 
//                                           global_y+halo, width_y,
//                                           p_x, width_x,
//                                           ps_y+halo, ps_x, tile_x_with_halo,
//                                           tid);
//       SM2REG<REAL,sizeof_rbuffer, LOCAL_TILE_Y, isBOX>(sm_rbuffer, r_smbuffer, 
//                                                     2*halo,
//                                                     ps_x, tid,
//                                                     tile_x_with_halo,
//                                                     2*halo);
//       REAL sum[LOCAL_TILE_Y];
//       init_reg_array<REAL,LOCAL_TILE_Y>(sum,0);
//       computation<REAL,LOCAL_TILE_Y,halo>(sum,
//                                       sm_rbuffer, ps_y, local_x+ps_x, tile_x_with_halo,
//                                       r_smbuffer, halo,
//                                       stencilParaInput);
//       reg2global<REAL,LOCAL_TILE_Y,LOCAL_TILE_Y>(sum, __var_4__, 
//                   global_y,p_y_cache, 
//                   p_x+local_x, width_x);
//       __syncthreads();
//       ptrselfcp<REAL,-halo, halo>(sm_rbuffer, ps_y, LOCAL_TILE_Y, tid, tile_x_with_halo);
//       REG2REG<REAL, sizeof_rbuffer, sizeof_rbuffer, 2*halo,isBOX>
//                 (r_smbuffer,r_smbuffer, LOCAL_TILE_Y, 0);
//     }
//     __syncthreads();
//     //computation of register space
//     if(UseRegCache)
//     {
//       _Pragma("unroll")
//       for(int local_y=0; local_y<total_reg_tile_y; local_y+=LOCAL_TILE_Y)
//       {
//         //deal with ew boundary
//         _Pragma("unroll")
//         for(int l_y=tid; l_y<LOCAL_TILE_Y+isBOX; l_y+=blockDim.x)
//         {
//           _Pragma("unroll")
//           for(int l_x=0; l_x<halo; l_x++)
//           {
//             // east
//             sm_rbuffer[(l_y+ps_y)*tile_x_with_halo+ tile_x + ps_x + l_x]=boundary_buffer[e_step + l_y + local_y + l_x * boundary_line_size];
//             // west
//             sm_rbuffer[(l_y+ps_y)*tile_x_with_halo+(-halo) + ps_x + l_x]=boundary_buffer[w_step + l_y + local_y + l_x * boundary_line_size];
//           }
//         }
//         reg2sm<REAL, sizeof_rspace, LOCAL_TILE_Y>(r_space, sm_rbuffer, 
//                                   ps_y+halo, ps_x, tid, 
//                                   tile_x_with_halo, local_y+halo*2);
//         __syncthreads();
//         SM2REG<REAL,sizeof_rbuffer, LOCAL_TILE_Y,isBOX>(sm_rbuffer, r_smbuffer, 
//                                                     2*halo,
//                                                     ps_x, tid,
//                                                     tile_x_with_halo,
//                                                     2*halo); 
//         REAL sum[LOCAL_TILE_Y];
//         init_reg_array<REAL,LOCAL_TILE_Y>(sum,0); 
//         computation<REAL,LOCAL_TILE_Y,halo>(sum,
//                                       sm_rbuffer, ps_y, local_x+ps_x, tile_x_with_halo,
//                                       r_smbuffer, halo,
//                                       stencilParaInput);
//         __syncthreads();
//         reg2reg<REAL, LOCAL_TILE_Y, sizeof_rspace, LOCAL_TILE_Y>(sum,r_space, 0, local_y);
//         ptrselfcp<REAL,-halo, halo>(sm_rbuffer, ps_y, LOCAL_TILE_Y, tid, tile_x_with_halo);
//         REG2REG<REAL, sizeof_rbuffer, sizeof_rbuffer, 2*halo,isBOX>
//                 (r_smbuffer,r_smbuffer, LOCAL_TILE_Y, 0);
//       }
//     }
//     if(UseSMCache)
//     //computation of share memory space
//     {
//       //load shared memory boundary
//       for(int local_y=tid; local_y<total_sm_tile_y; local_y+=blockDim.x)
//       {
//         // _Pragma("unroll")
//         for(int l_x=0; l_x<halo; l_x++)
//         {
//           // east
//           sm_space[(ps_y + local_y)*tile_x_with_halo+ tile_x + ps_x+l_x] = boundary_buffer[e_step + local_y + total_reg_tile_y + l_x*boundary_line_size];
//           //west
//           sm_space[(ps_y + local_y)*tile_x_with_halo+(-halo) + ps_x+l_x] = boundary_buffer[w_step + local_y + total_reg_tile_y + l_x*boundary_line_size];
//         }
//       }
//       __syncthreads();
//       //computation of shared space 
//       for ( size_t local_y = 0; local_y < total_sm_tile_y; local_y+=LOCAL_TILE_Y) 
//       {
//         SM2REG<REAL,sizeof_rbuffer,LOCAL_TILE_Y,isBOX>(sm_space, r_smbuffer, 
//                                           ps_y+local_y+halo, 
//                                           ps_x, tid,
//                                           tile_x_with_halo,
//                                           halo*2);
//         REAL sum[LOCAL_TILE_Y];
//         init_reg_array<REAL,LOCAL_TILE_Y>(sum,0);
        
//         computation<REAL,LOCAL_TILE_Y,halo>(sum,
//                                     sm_space, ps_y+local_y, local_x+ps_x, tile_x_with_halo,
//                                     r_smbuffer, halo,
//                                     stencilParaInput);
//         __syncthreads();
//         reg2sm<REAL, LOCAL_TILE_Y, LOCAL_TILE_Y>(sum, sm_space,
//                                     ps_y+local_y,
//                                     ps_x, tid,
//                                     tile_x_with_halo,
//                                     0);
//         __syncthreads();
//         REG2REG<REAL, sizeof_rbuffer, sizeof_rbuffer, 2*halo,isBOX>
//                 (r_smbuffer,r_smbuffer, LOCAL_TILE_Y, 0);
//       }
//     }
    
//     if(iter==iteration-1)break;
//     //register memory related boundary
//     //south
//     //*******************
//     if(UseRegCache)
//     {
//       if(tid>=blockDim.x-halo)
//       {
//         int l_x=tid-blockDim.x+halo;
//         //east
//         // #pragma unroll
//         _Pragma("unroll")
//         for(int l_y=0; l_y<total_reg_tile_y; l_y++)
//         {
//           boundary_buffer[e_step + l_y + l_x*boundary_line_size] = r_space[l_y];//sm_space[(ps_y + local_y) * BASIC_TILE_X + TILE_X + ps_x-Halo+0];
//         }
//       }
//       else if(tid<halo)
//       {
//         int l_x=tid;
//         //west
//         // #pragma unroll
//         _Pragma("unroll")
//         for(int l_y=0; l_y<total_reg_tile_y; l_y++)
//         {
//           boundary_buffer[w_step + l_y + l_x*boundary_line_size] = r_space[l_y];//sm_space[(ps_y + local_y) * BASIC_TILE_X + TILE_X + ps_x-Halo+0];
//         }
//       }
//     }
//     //store sm related boundary
//     if(UseSMCache)
//     {
//       _Pragma("unroll")
//       for(int local_y=tid; local_y<total_sm_tile_y; local_y+=blockDim.x)
//       {
//         _Pragma("unroll")
//         for(int l_x=0; l_x<halo; l_x++)
//         {
//           //east
//           boundary_buffer[e_step+local_y+total_reg_tile_y + l_x*boundary_line_size] = sm_space[(ps_y + local_y) * tile_x_with_halo + tile_x + ps_x - halo + l_x];
//           //west
//           boundary_buffer[w_step+local_y+total_reg_tile_y + l_x*boundary_line_size] = sm_space[(ps_y + local_y) * tile_x_with_halo + ps_x + l_x];
//         }
//       }
//     }
//     //deal with sm related boundary
//     //*******************
//     //store boundary to global (NS)
//     if(UseRegCache||UseSMCache)
//     {
//       _Pragma("unroll")
//       for(int l_y=0; l_y<halo; l_y++)
//       {
//         //north
//         if(UseSMCache)
//         {
//           __var_4__[(p_y_cache+(total_sm_tile_y+total_reg_tile_y)-halo+l_y) * width_x + p_x + tid]=sm_space[(ps_y + total_sm_tile_y - halo+l_y) * tile_x_with_halo + tid + ps_x];//boundary_buffer[N_STEP+tid+l_y*TILE_X];//
//         }
//         else
//         {
//           __var_4__[(p_y_cache+(total_sm_tile_y+total_reg_tile_y)-halo+l_y) * width_x + p_x + tid]=r_space[l_y+total_reg_tile_y-halo];
//         }
//          //south
//         if(UseRegCache)
//         {
//           __var_4__[(p_y_cache+l_y) * width_x + p_x + tid]= r_space[l_y];
//         }
//         else
//         {
//           __var_4__[(p_y_cache+l_y) * width_x + p_x + tid]= sm_space[(ps_y + l_y) * tile_x_with_halo + tid + ps_x];
//         }
//       }
//     }
//     //*******************
//     //store register part boundary
//     __syncthreads();
//     // store the whole boundary space to l2 cache
//     if(UseSMCache||UseRegCache)
//     {
//       _Pragma("unroll")
//       for(int lid=tid; lid<boundary_line_size-isBOX; lid+=blockDim.x)
//       {
//         //east
//         _Pragma("unroll")
//         for(int l_x=0; l_x<halo; l_x++)
//         {
//            //east
//           l2_cache_o[(((blockIdx.x* 2 + 1 )* halo+l_x)*width_y)  + p_y_cache +lid] = boundary_buffer[e_step+lid +l_x*boundary_line_size];
//           //west
//           l2_cache_o[(((blockIdx.x* 2 + 0) * halo+l_x)*width_y)  + p_y_cache +lid] = boundary_buffer[w_step+lid+l_x*boundary_line_size];
//         }
//       }
//     }
//     gg.sync();

//     REAL* tmp_ptr=__var_4__;
//     __var_4__=input;
//     input=tmp_ptr;

//     if(UseRegCache||UseSMCache)
//     {
//       tmp_ptr=l2_cache_o;
//       l2_cache_o=l2_cache_i;
//       l2_cache_i=tmp_ptr;
    
//       _Pragma("unroll")
//       for(int local_y=tid; local_y<boundary_line_size-isBOX; local_y+=blockDim.x)
//       {
//         _Pragma("unroll")
//         for(int l_x=0; l_x<halo; l_x++)
//         {
//           int cache_y=min(p_y_cache + local_y,width_y-1);
//           // east
//            boundary_buffer[e_step+local_y+l_x*boundary_line_size] = ((blockIdx.x == gridDim.x-1)?boundary_buffer[e_step+local_y+(halo-1)*boundary_line_size]:
//              l2_cache_i[(((blockIdx.x+1)*2+0)* halo+l_x)*width_y + cache_y]);
//            //west
//            boundary_buffer[w_step+local_y+l_x*boundary_line_size] = ((blockIdx.x == 0)?boundary_buffer[w_step+local_y+0*boundary_line_size]:
//             l2_cache_i[(((blockIdx.x-1)*2+1)* halo+l_x)*width_y + cache_y]);
//         }
//       }
//       for(int local_y=tid; local_y<isBOX&&p_y_cache + local_y + boundary_line_size-isBOX<width_y; local_y+=blockDim.x)
//       {
//         for(int l_x=0; l_x<halo; l_x++)
//         {
//           //east
//           int global_x = p_x + tile_x + l_x;
//           global_x = MIN(width_x-1,global_x);
//           boundary_buffer[e_step+local_y +boundary_line_size-isBOX+ l_x*boundary_line_size] = input[(p_y_cache + local_y+boundary_line_size-isBOX) * width_x + global_x];
//           //west
//           global_x = p_x - halo + l_x;
//           global_x = MAX(0,global_x);
//           boundary_buffer[w_step+local_y +boundary_line_size-isBOX+ l_x*boundary_line_size] =  input[(p_y_cache + local_y+boundary_line_size-isBOX) * width_x + global_x];
//         }
//       }
//     }

//     if(UseRegCache)
//     {
//       _Pragma("unroll")
//       for(int l_y=total_reg_tile_y-1; l_y>=0; l_y--)
//       {
//         r_space[l_y+halo]=r_space[l_y];
//       }
//     }

//   }

//   if(UseRegCache)
//   {
//     // register->global
//     reg2global<REAL, sizeof_rspace, total_reg_tile_y, false>(r_space, __var_4__,
//                                       p_y_cache, width_y,
//                                       p_x+tid, width_x,
//                                       0);
//   }
  
//   if(UseSMCache)
//   {
//     __syncthreads();
//     // shared memory -> global
//     sm2global<REAL,false>(sm_space, __var_4__, 
//                                     total_sm_tile_y,
//                                     p_y_cache+total_reg_tile_y, width_y,
//                                     p_x, width_x,
//                                     ps_y, ps_x, tile_x_with_halo ,
//                                     tid);
//   }
//   #undef UseRegCache
//   #undef SM2REG
//   #undef REG2REG
// }
// #endif

__global__ void printptx()
{
  printf("code is run in %d\n",PERKS_ARCH);
}
void host_printptx()
{
  printptx<<<1,1>>>();
  hipDeviceSynchronize();
}

#ifndef RTILE_Y
#define RTILE_Y (8)
#endif
#ifndef TILE_X
#define TILE_X (256)
#endif

#define bdim_x (TILE_X)

#define BASIC_TILE_X (TILE_X+2*Halo)
#define BASIC_TILE_Y (RTILE_Y+2*Halo)
#define BASIC_SM_SPACE (BASIC_TILE_X)*(BASIC_TILE_Y)


#define TOTAL_SM_TILE_Y (RTILE_Y*SM_FOLER_Y)
#define TOTAL_REG_TILE_Y (RTILE_Y*REG_FOLDER_Y)
#define TOTAL_SM_CACHE_SPACE (TILE_X+2*Halo)*(TOTAL_SM_TILE_Y+2*Halo)

#define TILE_Y (TOTAL_SM_TILE_Y+TOTAL_REG_TILE_Y)




template<class REAL>
void jacobi_iterative(REAL * h_input, int width_y, int width_x, REAL * __var_0__, int iteration){
// extern "C" void jacobi_iterative(REAL * h_input, int width_y, int width_x, REAL * __var_0__, int iteration){
/* Host allocation Begin */
  host_printptx();
/*************************************/


//initialization
#if defined(PERSISTENT)
  auto execute_kernel = kernel_persistent_baseline<REAL,RTILE_Y,Halo>;
#endif
#if defined(BASELINE_CM)||defined(BASELINE)
  #ifndef BOX
    auto execute_kernel = kernel_baseline<REAL,RTILE_Y,Halo>;
  #else
    auto execute_kernel = kernel_baseline_box<REAL,RTILE_Y,Halo>;
  #endif
#endif
#ifdef NAIVE
  #ifndef BOX
    auto execute_kernel = kernel2d_restrict<REAL,Halo>;
  #else
    auto execute_kernel = kernel2d_restrict_box<REAL,Halo>;
  #endif
#endif 
#ifdef GEN
  auto execute_kernel = kernel_general<REAL,RTILE_Y,Halo,REG_FOLDER_Y,true>;
  //auto execute_kernel = kernel_general<REAL,RTILE_Y,Halo,REG_FOLDER_Y,UseSMCache>;
#endif
  int sm_count;
  hipDeviceGetAttribute ( &sm_count, hipDeviceAttributeMultiprocessorCount,0 );
  
  //initialization input and output space
  REAL * input;
  hipMalloc(&input,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : input\n");
  hipMemcpy(input,h_input,sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyHostToDevice);
  REAL * __var_1__;
  hipMalloc(&__var_1__,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  REAL * __var_2__;
  hipMalloc(&__var_2__,sizeof(REAL)*((width_y-0)*(width_x-0)));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");

  //initialize tmp space for halo region
#if defined(GEN) || defined(MIX)|| defined(PERSISTENT)
  REAL * L2_cache3;
  REAL * L2_cache4;
  size_t L2_utage_2 = sizeof(REAL)*(width_y)*2*(width_x/bdim_x)*Halo;
#ifndef __PRINT__
  printf("l2 cache used is %ld KB : 4096 KB \n",L2_utage_2*2/1024);
#endif
  hipMalloc(&L2_cache3,L2_utage_2*2);
  L2_cache4=L2_cache3+(width_y)*2*(width_x/bdim_x)*Halo;
#endif

  //initialize shared memory
  int maxSharedMemory;
  hipDeviceGetAttribute (&maxSharedMemory, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,0 );
  //could not use all share memory in a100. so set it in default.
  int SharedMemoryUsed = maxSharedMemory-1024;

#if defined(USEMAXSM)
  hipFuncSetAttribute(reinterpret_cast<const void*>(execute_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, SharedMemoryUsed);
#endif

size_t executeSM = 0;
#ifndef NAIVE
  //shared memory used for compuation
  int basic_sm_space=(RTILE_Y+2*Halo)*(TILE_X+2*Halo);
  size_t sharememory_basic=(1+basic_sm_space)*sizeof(REAL);
  executeSM = sharememory_basic;

#endif

  #ifdef PERSISTENT
    size_t max_sm_flder=0;
  #endif 

  #define halo Halo
  #if defined(GEN) || defined(MIX)
  size_t max_sm_flder=0;
  max_sm_flder=(SharedMemoryUsed/sizeof(REAL)
                          -2*Halo*isBOX
                          -basic_sm_space
                          -2*Halo*(REG_FOLDER_Y)*RTILE_Y
                          -2*Halo*(TILE_X+2*Halo))/(TILE_X+4*Halo)/RTILE_Y;

  // size_t sm_cache_size = TOTAL_SM_CACHE_SPACE*sizeof(REAL);
  size_t sm_cache_size = (max_sm_flder*RTILE_Y+2*Halo)*(TILE_X+2*Halo)*sizeof(REAL);
  size_t y_axle_halo = (Halo*2*((max_sm_flder + REG_FOLDER_Y)*RTILE_Y+isBOX))*sizeof(REAL);
  executeSM=sharememory_basic+y_axle_halo;
  executeSM+=sm_cache_size;
  #undef halo
#ifndef __PRINT__
  printf("the max flder is %ld and the total sm size is %ld\n", max_sm_flder, executeSM);
#endif

  //size_t sharememory3=sharememory_basic+(Halo*2*(TILE_Y))*sizeof(REAL);
  //size_t sharememory4=sharememory3-(STILE_SIZE*sizeof(REAL));
#endif


#ifdef PERSISTENTTHREAD
  int numBlocksPerSm_current=0;

  #ifdef MIX
    if(SM_FOLER_Y!=0)
    {
      // hipLaunchCooperativeKernel((void*)kernel_mix, grid_dim, block_dim, KernelArgs2,sharememory3,0);
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, kernel_mix, bdim_x, sharememory3);
    }
    else
    {
      // hipLaunchCooperativeKernel((void*)kernel_mix_reg, grid_dim, block_dim, KernelArgs2,sharememory4,0);
      hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, kernel_mix_reg, bdim_x, sharememory4);
    }
  
  #endif
  #if defined(BASELINE_CM)||defined(PERSISTENT)||defined(GEN)
    hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, execute_kernel, bdim_x, executeSM);
  #endif

  dim3 block_dim(bdim_x);
  dim3 grid_dim(width_x/bdim_x,sm_count*numBlocksPerSm_current/(width_x/bdim_x));
  
  dim3 executeBlockDim=block_dim;
  dim3 executeGridDim=grid_dim;
#endif 
#ifdef NAIVE
  dim3 block_dim_1(MIN(width_x,bdim_x),1);
  dim3 grid_dim_1(width_x/MIN(width_x,bdim_x),width_y/1);

  dim3 executeBlockDim=block_dim_1;
  dim3 executeGridDim=grid_dim_1;
#endif
#ifdef BASELINE
  dim3 block_dim2(bdim_x);
  dim3 grid_dim2(width_x/bdim_x,MIN((sm_count*8*1024/bdim_x)/(width_x/bdim_x),width_y/RTILE_Y));
//  printf("<%d,%d,%d>",); 
  dim3 executeBlockDim=block_dim2;
  dim3 executeGridDim=grid_dim2;

#endif
//in order to get a better performance, warmup run is necessary.

#ifdef MIX
  int l_iteration=iteration;
  void* KernelArgs2[] ={(void**)&input,(void**)&width_y,
    (void*)&width_x,(void*)&__var_2__,(void*)&L2_cache1,(void*)&L2_cache1,
    (void*)&l_iteration};
#endif

#if defined(GEN) || defined(PERSISTENT)
  int l_iteration=iteration;
  void* ExecuteKernelArgs[] ={(void**)&input,(void**)&width_y,
    (void*)&width_x,(void*)&__var_2__,(void*)&L2_cache3,(void*)&L2_cache4,
    (void*)&l_iteration, (void*)&max_sm_flder};

  #ifdef WARMUPRUN
    void* KernelArgs_NULL[] ={(void**)&__var_2__,(void**)&width_y,
      (void*)&width_x,(void*)&__var_1__,(void*)&L2_cache3,(void*)&L2_cache4,
      (void*)&l_iteration, (void *)&max_sm_flder};
  #endif

#endif

#if defined(GEN) && defined(L2PER)
    REAL l2perused;
    size_t inner_window_size = 30*1024*1024;
    hipLaunchAttributeValue stream_attribute;
    stream_attribute.accessPolicyWindow.base_ptr  = reinterpret_cast<void*>(L2_cache3);                  // Global Memory data pointer
    stream_attribute.accessPolicyWindow.num_bytes = min(inner_window_size,L2_utage_2*2);                                   // Number of bytes for persistence access
    stream_attribute.accessPolicyWindow.hitRatio  = 1;                                             // Hint for cache hit ratio
    stream_attribute.accessPolicyWindow.hitProp   = hipAccessPropertyPersisting;                  // Persistence Property
    stream_attribute.accessPolicyWindow.missProp  = hipAccessPropertyStreaming;  

    hipStreamSetAttribute(0, hipLaunchAttributeAccessPolicyWindow, &stream_attribute); 
    cudaCtxResetPersistingL2Cache();
    hipStreamSynchronize(0);
#endif

#ifdef WARMUPRUN
      cudaCheckError();
  #ifdef TRADITIONLAUNCH
      execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (__var_2__, width_y, width_x,__var_1__);
  #endif 

  #ifdef PERSISTENTLAUNCH
      hipLaunchCooperativeKernel((void*)execute_kernel, executeGridDim, executeBlockDim, KernelArgs_NULL, executeSM,0);
  #endif

#endif 

#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif
#ifdef MIX
  if(SM_FOLER_Y!=0)
  {
    hipLaunchCooperativeKernel((void*)kernel_mix, grid_dim, block_dim, KernelArgs2,sharememory3,0);
  }
  else
  {
    hipLaunchCooperativeKernel((void*)kernel_mix_reg, grid_dim, block_dim, KernelArgs2,sharememory4,0);
  }
#endif
#ifdef PERSISTENTLAUNCH
  hipLaunchCooperativeKernel((void*)execute_kernel, 
            executeGridDim, executeBlockDim, 
            ExecuteKernelArgs, 
            //KernelArgs4,
            executeSM,0);
#endif
#ifdef TRADITIONLAUNCH
  execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
          (input, width_y, width_x, __var_2__);

  for(int i=1; i<iteration; i++)
  {
     execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
          (__var_2__, width_y, width_x , __var_1__);
    REAL* tmp = __var_2__;
    __var_2__=__var_1__;
    __var_1__= tmp;
  }
  cudaCheckError();
#endif


#ifdef CHECK
  hipDeviceSynchronize();
  cudaCheckError();
#endif

#ifndef __PRINT__  
  printf("sm_count is %d\n",sm_count);
  printf("MAX shared memory is %f KB but only use %f KB\n",maxSharedMemory/1024.0,SharedMemoryUsed/1024.0);
  printf(" shared meomory size is %ld KB\n", executeSM/1024);

#endif

#ifdef __PRINT__
  #ifdef BASELINE
    #ifndef DA100X
      printf("bsln\t");
    #else
      printf("asyncbsln\t");
    #endif
  #endif 
  #ifdef BASELINE_CM
    #ifndef DA100X
      printf("bsln_cm\t");
    #else
      printf("asyncbsln_cm\t");
    #endif
  #endif 
  
  #ifdef NAIVE
    printf("naive\t");
  #endif 

  #ifdef PERSISTENT
    #ifndef DA100X
      printf("psstnt\t");
    #else
      printf("asyncpsstnt\t");
    #endif
  #endif

  // #ifdef GEN
  //     printf("gen"); 
  //   #else
  //     printf("asyncgen"); 
  //   #endif
  //   #if REG_FOLDER_Y==0 && SM_FOLER_Y ==0
  //     printf("\t");
  //   #endif
  //   #if REG_FOLDER_Y==0 && SM_FOLER_Y !=0
  //     printf("_sm\t");
  //   #endif
  //   #if REG_FOLDER_Y!=0 && SM_FOLER_Y ==0
  //     printf("_reg\t");
  //   #endif
  //   #if REG_FOLDER_Y!=0 && SM_FOLER_Y !=0
  //     printf("_mix\t");
  //   #endif
  // #endif
#endif 

#ifdef __PRINT__
  printf("%d\t%d\t%d\t",width_x,width_y,iteration);
  printf("<%d,%d>\t<%d,%d>\t%d\t0\t0\t",executeBlockDim.x,1,
        executeGridDim.x,executeGridDim.y,
        executeGridDim.x*executeGridDim.y/sm_count);
#endif

#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
  #ifdef __PRINT__
  printf("%f\t%f\n",elapsedTime,(REAL)iteration*(width_y)*(width_x)/ elapsedTime/1000/1000);
  #else
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  printf("[FORMA] Speed(GCells/s) : %lf\n",(REAL)iteration*(width_y)*(width_x)/ elapsedTime/1000/1000);
  printf("[FORMA] Speed(GFLOPS/s) : %lf\n", (REAL)17*iteration*(width_y)*(width_x)/ elapsedTime/1000/1000);
  printf("[FORMA] bandwidth(GB/s) : %lf\n", (REAL)sizeof(REAL)*iteration*((width_y)*(width_x)+width_x*width_y)/ elapsedTime/1000/1000);
  printf("[FORMA] width_x:width_y=%d:%d\n",(int)width_x, (int)width_y);
#if defined(GEN) || defined(MIX)
  printf("[FORMA] cached width_x:width_y=%d:%d\n",(int)TILE_X*grid_dim.x, (int)(max_sm_flder+REG_FOLDER_Y)*RTILE_Y*grid_dim.y);
  printf("[FORMA] cached b:sf:rf=%d:%d:%d\n", (int)RTILE_Y, (int)max_sm_flder, (int)REG_FOLDER_Y);
#endif
  #endif

  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif


//finalization
#ifdef CHECK
  // printf("check error here*\n");
  hipDeviceSynchronize();
  cudaCheckError();
#endif

#if defined(GEN) || defined(PERSISTENT)
  if(iteration%2==1)
    hipMemcpy(__var_0__,__var_2__, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
  else
    hipMemcpy(__var_0__,input, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
#else
  hipMemcpy(__var_0__,__var_2__, sizeof(REAL)*((width_y-0)*(width_x-0)), hipMemcpyDeviceToHost);
#endif
/*Kernel Launch End */
/* Host Free Begin */
  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);

  // hipFree(L2_cache);
  // hipFree(L2_cache1);
  // hipFree(L2_cache2);
#if defined(GEN) || defined(PERSISTENT)
  hipFree(L2_cache3);
#endif
  // hipFree(L2_cache4);

}

PERKS_INITIALIZE_ALL_TYPE(PERKS_DECLARE_INITIONIZATION_ITERATIVE);


