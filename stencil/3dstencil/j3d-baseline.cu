#include "hip/hip_runtime.h"
#include "./config.cuh"

#include "./common/cuda_computation.cuh"
#include "./common/cuda_common.cuh"
#include "./common/types.hpp"
#include <math.h>

#include <hip/hip_cooperative_groups.h>

#ifdef ASYNCSM
  // #if PERKS_ARCH<800 
    // #error "unsupport architecture"
  // #endif
  #include <cooperative_groups/memcpy_async.h>
  #include <cuda_pipeline.h>
#endif

namespace cg = cooperative_groups;

template<class REAL, int halo>
__global__ void 
#ifndef PERSISTENT
kernel3d_baseline(REAL * __restrict__ input, 
                                REAL * __restrict__ output, 
                                int width_z, int width_y, int width_x) 
#else
kernel3d_persistent(REAL * __restrict__ input, 
                                REAL * __restrict__ output, 
                                int width_z, int width_y, int width_x,
                                REAL* l2_cache_i, REAL* l2_cache_o,
                                int iteration) 
#endif
{
  // printf("?");
  const int tile_x_with_halo=TILE_X+2*halo;
  const int tile_y_with_halo=TILE_Y+2*halo;
  stencilParaT;
  
  extern __shared__ char sm[];
  REAL* sm_rbuffer = (REAL*)sm+1;

  register REAL r_smbuffer[2*halo+1][ITEM_PER_THREAD];
  // printf("%d\n",ITEM_PER_THREAD);
  // return;
  REAL* smbuffer_buffer_ptr[halo+1];
  smbuffer_buffer_ptr[0]=sm_rbuffer;
  #pragma unroll
  for(int hl=1; hl<halo+1; hl++)
  {
    smbuffer_buffer_ptr[hl]=smbuffer_buffer_ptr[hl-1]+tile_x_with_halo*tile_y_with_halo;
  }

  const int tid_x = threadIdx.x%TILE_X;
  const int tid_y = threadIdx.x/TILE_X;
  const int dim_y = TILE_Y/ITEM_PER_THREAD;

  const int cpblocksize_y=(TILE_Y+2*halo)/dim_y;
  const int cpquotion_y=(TILE_Y+2*halo)%dim_y;

  const int index_y = ITEM_PER_THREAD*tid_y;

  const int cpbase_y = -halo+tid_y*cpblocksize_y+(tid_y<=cpquotion_y?tid_y:cpquotion_y);
  const int cpend_y = cpbase_y + cpblocksize_y + (tid_y<=cpquotion_y?1:0);
  // if(tid_x==0&&blockIdx.x==0)
  //   printf("<%d,%d,%d,%d,%d,%d>",cpbase_y,cpend_y,cpblocksize_y,dim_y,tid_y,ITEM_PER_THREAD);
  // return;
  const int ps_y = halo;
  const int ps_x = halo;
  // const int ps_z = halo;

  const int p_x = blockIdx.x * TILE_X;
  const int p_y = blockIdx.y * TILE_Y;
  // if(blockIdx.x==0&&tid_x==0)
  //   printf("<%d,%d,%d:%d,%d>",ps_x,ps_y,ps_z,tid_x,tid_y);
  // return;
  int blocksize_z=(width_z/gridDim.z);
  int z_quotient = width_z%gridDim.z;

  const int p_z =  blockIdx.z * (blocksize_z) + (blockIdx.z<=z_quotient?blockIdx.z:z_quotient);
  blocksize_z += (blockIdx.z<z_quotient?1:0);
  const int p_z_end = p_z + (blocksize_z);
 
  // int smz_ind=0;
#ifdef PERSISTENT  
  cg::grid_group gg = cg::this_grid();
  for(int iter=0; iter<iteration; iter++)
#endif
  {
    global2regs3d<REAL, ITEM_PER_THREAD, 1+2*halo>
      (input, r_smbuffer, p_z-halo,width_z, p_y+index_y, width_y, p_x, width_x,tid_x);
    //need optimization to remove redundent memory access
    global2sm<REAL, halo, 0, halo, halo+1, 0, true, false>
                                        (input, smbuffer_buffer_ptr,
                                          p_x, p_y, p_z,
                                          width_x, width_y, width_z,

                                          tile_x_with_halo, ps_x,
                                          // -halo+index_y, -halo+index_y+ITEM_PER_THREAD+2*halo, ps_y,
                                          cpbase_y, cpend_y,ps_y,
                                          // cpsize_y, ps_y,cpbase_y, 
                                          TILE_X, tid_x);
    __syncthreads();
    for(int global_z=p_z; global_z<p_z_end; global_z+=1)
    {
      __syncthreads();
      
      global2sm<REAL, halo, halo, 1, halo+1, 0, false, true>
                                        (input, smbuffer_buffer_ptr,
                                          p_x, p_y, global_z,
                                          width_x, width_y, width_z,

                                          tile_x_with_halo, ps_x,
                                          // -halo+index_y, -halo+index_y+ITEM_PER_THREAD+2*halo, ps_y,
                                          cpbase_y, cpend_y,ps_y,
                                          TILE_X,tid_x);
      __syncthreads();
      

      //sm2reg

      sm2regs<REAL, ITEM_PER_THREAD, 1+2*halo, 
                1+halo, halo, 
                0, halo*2, 
                ITEM_PER_THREAD, 1>
        (smbuffer_buffer_ptr, r_smbuffer, 
          ps_y+index_y, ps_x, 
          tile_x_with_halo, tid_x);

      REAL sum[ITEM_PER_THREAD];
      // #pragma unroll
      _Pragma("unroll")
      for(int l_y=0; l_y<ITEM_PER_THREAD; l_y++)
      {
        sum[l_y]=0;
      }

      //main computation
      computation<REAL,ITEM_PER_THREAD,halo>( sum,
                                      smbuffer_buffer_ptr[0],
                                      ps_y+index_y, tile_x_with_halo, tid_x+ps_x,
                                      r_smbuffer,
                                      stencilParaInput);

      // // reg 2 ptr
      reg2global3d<REAL, ITEM_PER_THREAD>(
            sum, output,
            global_z, width_z,
            p_y+index_y, width_y,
            p_x, width_x,
            tid_x);

      REAL* tmp = smbuffer_buffer_ptr[0];
      // smswap 
      _Pragma("unroll")
      for(int hl=1; hl<halo+1; hl++)
      {
        smbuffer_buffer_ptr[hl-1]=smbuffer_buffer_ptr[hl];
      }
      smbuffer_buffer_ptr[halo]=tmp;

      regsself3d<REAL,2*halo+1,ITEM_PER_THREAD>(r_smbuffer);

    }
    #ifdef PERSISTENT
      if(iter>=iteration-1)break;
      // if(threadIdx.x==0&&blockIdx.x==0&&blockIdx.y==0&&blockDim.x==0)
      // {
      //   printf("%d\n",iter);
      // }
      gg.sync();

      REAL* tmp_ptr =output;
      output=input;
      input=tmp_ptr;
    #endif
  }
}


// template __global__ void kernel3d_baseline<float,HALO> 
//     (float *__restrict__, float *__restrict__ , int , int , int );
// template __global__ void kernel3d_baseline<double,HALO> 
//     (double *__restrict__, double *__restrict__ , int , int , int );

#ifndef PERSISTENT 
  PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_BASELINE,HALO);
#else
  PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_PERSISTENT,HALO);
#endif