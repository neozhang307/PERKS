#include "hip/hip_runtime.h"
#ifndef CONFIGURE
  #include "./config.cuh"
  #include "./genconfig.cuh"
#endif
#include "./common/cuda_computation.cuh"
#include "./common/cuda_common.cuh"
#include "./common/types.hpp"
#include <math.h>

#include <hip/hip_cooperative_groups.h>

#ifdef ASYNCSM
  // #if PERKS_ARCH<800 
    // #error "unsupport architecture"
  // #endif
  #include <cooperative_groups/memcpy_async.h>
  #include <cuda_pipeline.h>
#endif

#ifdef USESM
#define isUseSM (true)
#else
#define isUseSM (false)
#endif
namespace cg = cooperative_groups;

// #define NOCACHE_Y (0)
#define NOCACHE_Z (HALO)
// #define LOCAL_TILE_Y (TILE_Y-2*NOCACHE_Y)
#include "./j3d-general-kernels.cuh"


#define MAXTHREAD (256)
#define MINBLOCK (1)

template<class REAL, int halo, 
int LOCAL_ITEM_PER_THREAD, int LOCAL_TILE_X, int LOCAL_TILE_Y, const int reg_folder_z,int minblocks, bool UseSMCache >
// __launch_bounds__(256, 2)
__launch_bounds__(MAXTHREAD, minblocks)
__global__ void 
kernel3d_general(REAL * __restrict__ input, 
                                REAL * __restrict__ output, 
                                int width_z, int width_y, int width_x,
                                REAL* l2_cache_i, REAL* l2_cache_o,
                                int iteration,
                                int max_sm_flder) 
{
  kernel3d_general_inner<REAL,halo,LOCAL_ITEM_PER_THREAD,LOCAL_TILE_X,LOCAL_TILE_Y,reg_folder_z,UseSMCache>
  (input,output,width_z,width_y,width_x,l2_cache_i,l2_cache_o,iteration,max_sm_flder);
}


// template __global__ void kernel3d_general<float,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,0,false> 
//     (float *__restrict__, float *__restrict__ , int , int , int, float*,float*,int,int);

// template __global__ void kernel3d_general<float,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,0,true> 
//     (float *__restrict__, float *__restrict__ , int , int , int, float*,float*,int,int);

// template __global__ void kernel3d_general<float,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,false> 
//     (float *__restrict__, float *__restrict__ , int , int , int, float*,float*,int,int);

// template __global__ void kernel3d_general<float,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,true> 
//     (float *__restrict__, float *__restrict__ , int , int , int, float*,float*,int,int);

// template __global__ void kernel3d_general<double,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,true> 
//     (double *__restrict__, double *__restrict__ , int , int , int , double*,double*,int,int);
#ifndef CONFIGURE
  PERKS_INITIALIZE_ALL_TYPE_7ARG(PERKS_DECLARE_INITIONIZATION_GENERAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,MINBLOCK,true);
  PERKS_INITIALIZE_ALL_TYPE_7ARG(PERKS_DECLARE_INITIONIZATION_GENERAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,MINBLOCK,false);
#else
  template __global__ void kernel3d_general<TYPE,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,BLOCKTYPE,isUseSM> 
    (TYPE *__restrict__, TYPE *__restrict__ , int , int , int, TYPE*,TYPE*,int,int);
#endif
// #ifndef PERSISTENT 
  // PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_BASELINE,HALO);
// #else
  // PERKS_INITIALIZE_ALL_TYPE_1ARG(PERKS_DECLARE_INITIONIZATION_PERSISTENT,HALO);
// #endif