#include "hip/hip_runtime.h"
// #include "./common/common.hpp"
// #include <hip/hip_cooperative_groups.h>
// #include <hip/hip_runtime.h>
// #include "stdio.h"
// #include "./common/cuda_computation.cuh"
// #include "./common/cuda_common.cuh"
// #include "./common/types.hpp"
#include "genconfig.cuh"
#ifdef _TIMER_
#include "hip/hip_runtime_api.h"
#endif
#include <hip/hip_runtime.h>
#include "stdio.h"
#include <hip/hip_cooperative_groups.h>
#include "stdio.h"
#include "assert.h"
#include "config.cuh" 
#include "./common/jacobi_cuda.cuh"
#include "./common/types.hpp"
#include "./common/cuda_common.cuh"
#include "./common/cuda_computation.cuh"

// #define TILE_X 256
// #define NAIVE
#if defined(NAIVE)||defined(BASELINE)||defined(BASELINE_CM)
  #define TRADITIONLAUNCH
#endif
#if defined(GEN)||defined(PERSISTENT)
  #define PERSISTENTLAUNCH
#endif
#if defined PERSISTENTLAUNCH||defined(BASELINE_CM)
  #define PERSISTENTTHREAD
#endif
#if defined(BASELINE)||defined(BASELINE_CM) ||defined(GEN)||defined(GENWR)||defined(PERSISTENT)
  #define USEMAXSM
#endif


#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
 }                                                                 \
}


__global__ void printptx(int *result)
{
  // printf("code is run in %d\n",PERKS_ARCH);
  result[0]=PERKS_ARCH;
}
void host_printptx(int&result)
{
  int*d_r;
  hipMalloc((void**)&d_r, sizeof(int));
  printptx<<<1,1>>>(d_r);
  hipMemcpy(&result, d_r, sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}


template<class REAL>
int j3d_iterative(REAL * h_input,
  int height, int width_y, int width_x,
  REAL * __var_0__, 
  int global_bdimx, 
  int blkpsm, 
  int iteration, 
  bool useSM,
  bool usewarmup, 
  int warmupiteration)
{
  // int iteration=4;
/* Host allocation Begin */
  int sm_count;
  hipDeviceGetAttribute ( &sm_count, hipDeviceAttributeMultiprocessorCount,0 );
#ifndef __PRINT__
  printf("sm_count is %d\n",sm_count);
#endif

  int ptx;
  host_printptx(ptx);
#ifndef __PRINT__
  printf("code is run in %d\n",ptx);
#endif
#ifdef NAIVE
  auto execute_kernel = kernel3d_restrict<REAL,HALO>;
#endif 
#if defined(BASELINE) ||defined(BASELINE_CM)
  auto execute_kernel = kernel3d_baseline<REAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y>;
#endif
#ifdef PERSISTENT
  auto execute_kernel = kernel3d_persistent<REAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y>;
#endif
#ifdef GEN
  auto execute_kernel = useSM? kernel3d_general<REAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,1,true>
                        : kernel3d_general<REAL,HALO,ITEM_PER_THREAD,TILE_X,TILE_Y,REG_FOLDER_Z,1,false>;
#endif

//shared memory related 
size_t executeSM=0;
#ifndef NAIVE
    int basic_sm_space=((TILE_Y+2*HALO)*(TILE_X+2*HALO)*(1+HALO+isBOX)+1)*sizeof(REAL);
    executeSM=basic_sm_space;
#endif
// printf("sm is %ld\n",executeSM);
// #if defined(GEN) || defined(MIX)
    // int sharememory1 = basic_sm_space+2*BD_STEP_XY*FOLDER_Z*sizeof(REAL);
    // int sharememory2 = sharememory1 + sizeof(REAL) * (SFOLDER_Z)*(TILE_Y*2-1)*TILE_X;
// #endif

  REAL * input;
  hipMalloc(&input,sizeof(REAL)*(height*width_x*width_y));
  Check_CUDA_Error("Allocation Error!! : input\n");

  hipGetLastError();
  hipMemcpy(input,h_input,sizeof(REAL)*(height*width_x*width_y), hipMemcpyHostToDevice);
  REAL * __var_1__;
  hipMalloc(&__var_1__,sizeof(REAL)*(height*width_x*width_y));
  Check_CUDA_Error("Allocation Error!! : __var_1__\n");
  REAL * __var_2__;
  hipMalloc(&__var_2__,sizeof(REAL)*(height*width_x*width_y));
  Check_CUDA_Error("Allocation Error!! : __var_2__\n");

#ifdef USEMAXSM
  int maxSharedMemory;
  hipDeviceGetAttribute (&maxSharedMemory, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,0 );
  int SharedMemoryUsed=maxSharedMemory-1024;
  hipFuncSetAttribute(reinterpret_cast<const void*>(execute_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, SharedMemoryUsed);
#endif 

/*Host Allocation End */
/* Kernel Launch Begin */
// #ifndef

#ifdef NAIVE
  dim3 block_dim_1(bdimx, 4, 1);
  dim3 grid_dim_1(width_x/bdimx, width_y/4, height);

  dim3 executeBlockDim=block_dim_1;
  dim3 executeGridDim=grid_dim_1;

#endif
#ifdef BASELINE
  dim3 block_dim_2(bdimx, 1, 1);
  dim3 grid_dim_2(width_x/TILE_X, width_y/TILE_Y, max(2,(sm_count*8)*TILE_X*TILE_Y/width_x/width_y));
  // dim3 block_dim3(TILE_X, 1, 1);
  // dim3 grid_dim3(MIN(width_x*width_y/TILE_X/TILE_Y,sm_count*numBlocksPerSm_current), 1, sm_count*numBlocksPerSm_current/MIN(width_x*width_y/TILE_X/TILE_Y,sm_count*numBlocksPerSm_current));
  
  dim3 executeBlockDim=block_dim_2;
  dim3 executeGridDim=grid_dim_2;
#endif
// #ifdef BASELINE_MEMWARP
//   dim3 block_dim_2(bdimx+2*TILE_X, 1, 1);
//   dim3 grid_dim_2(width_x/TILE_X, width_y/TILE_Y,max(2,(sm_count*8)*TILE_X*TILE_Y/width_x/width_y));
//   // dim3 block_dim3(TILE_X, 1, 1);
//   // dim3 grid_dim3(MIN(width_x*width_y/TILE_X/TILE_Y,sm_count*numBlocksPerSm_current), 1, sm_count*numBlocksPerSm_current/MIN(width_x*width_y/TILE_X/TILE_Y,sm_count*numBlocksPerSm_current));
  
//   dim3 executeBlockDim=block_dim_2;
//   dim3 executeGridDim=grid_dim_2;
// #endif

#ifdef PERSISTENTLAUNCH
  int max_sm_flder=0;
#endif


#if defined(PERSISTENTTHREAD)
  int numBlocksPerSm_current=100;

  #if defined(GEN)
    int reg_folder_z=REG_FOLDER_Z;
    executeSM+=reg_folder_z*2*HALO*(TILE_Y+TILE_X+2*isBOX);
  #endif 
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, execute_kernel, bdimx, executeSM);
  cudaCheckError();
  if(blkpsm<=0)blkpsm=numBlocksPerSm_current;
  numBlocksPerSm_current=min(blkpsm,numBlocksPerSm_current);
  // numBlocksPerSm_current=1;
  dim3 block_dim_3(bdimx, 1, 1);
  dim3 grid_dim_3(width_x/TILE_X, width_y/TILE_Y, MAX(1,sm_count*numBlocksPerSm_current/(width_x*width_y/TILE_X/TILE_Y)));
  dim3 executeBlockDim=block_dim_3;
  dim3 executeGridDim=grid_dim_3;

  printf("plckpersm is %d\n", numBlocksPerSm_current);
  // printf("plckpersm is %f\n", (double)executeSM);
#endif

#if defined(GEN)

  // 
  int perSMUsable=SharedMemoryUsed/numBlocksPerSm_current;
  int perSMValsRemaind=(perSMUsable-basic_sm_space)/sizeof(REAL);
  int reg_boundary=reg_folder_z*2*HALO*(TILE_Y+TILE_X+2*isBOX);
  // assert(perSMValsRemaind>=reg_boundary);
  max_sm_flder=(perSMValsRemaind-reg_boundary)/(2*HALO*(TILE_Y+TILE_X*2*isBOX)+TILE_X*TILE_Y);
  // printf("<%d>\n",max_sm_flder);
  if(!useSM)max_sm_flder=0;
  if(useSM&&max_sm_flder==0)return 1;

  int sharememory1 = 2*HALO*(TILE_Y+TILE_X+2*isBOX)*(max_sm_flder+reg_folder_z)*sizeof(REAL);//boundary
  int sharememory2 = sharememory1 + sizeof(REAL) * (max_sm_flder)*(TILE_Y)*TILE_X;
  // printf("%d\n",executeSM);
  executeSM=sharememory2+basic_sm_space;
  printf("folder %d sm = %d\n",max_sm_flder,executeSM);

#endif
  // printf("<%d,%d,%d>",executeGridDim.x,executeGridDim.y,executeGridDim.z);

  size_t L2_utage = width_y*height*sizeof(REAL)*HALO*(width_x/TILE_X)*2+
                    width_x*height*sizeof(REAL)*HALO*(width_y/TILE_Y)*2  ;

  REAL * l2_cache1;
  REAL * l2_cache2;
  hipMalloc(&l2_cache1,L2_utage);
  hipMalloc(&l2_cache2,L2_utage);
#ifndef __PRINT__
  printf("l2 cache used is %ld KB : 4096 KB \n",L2_utage/1024);
#endif

  int l_warmupiteration=warmupiteration>0?warmupiteration:1000;

#ifdef PERSISTENTLAUNCH
  int l_iteration=iteration;
  void* KernelArgs[] ={(void**)&input,(void*)&__var_2__,
    (void**)&height,(void**)&width_y,(void*)&width_x,
    (void**)&l2_cache1, (void**)&l2_cache2,
    (void*)&l_iteration,(void*)&max_sm_flder};
  // #ifdef __PRINT__  
  void* KernelArgsNULL[] ={(void**)&__var_2__,(void*)&__var_1__,
      (void**)&height,(void**)&width_y,(void*)&width_x,
      (void**)&l2_cache1, (void**)&l2_cache2,
      (void*)&l_warmupiteration,(void*)&max_sm_flder};
  // #endif
#endif
cudaCheckError();
// bool warmup=false;
if(usewarmup)
{
  hipEvent_t warstart,warmstop;
  hipEventCreate(&warstart);
  hipEventCreate(&warmstop);
  #ifdef TRADITIONLAUNCH
  {
      hipEventRecord(warstart,0);
      // cudaCheckError();
      for(int i=0; i<l_warmupiteration; i++)
      {
        // execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
        //       (__var_2__, width_y, width_x , __var_1__);
        execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (__var_2__, __var_1__,  height, width_y, width_x);
        REAL* tmp = __var_2__;
        __var_2__=__var_1__;
        __var_1__= tmp;

      } 
      hipEventRecord(warmstop,0);
      hipEventSynchronize(warmstop);
      cudaCheckError();
      float warmelapsedTime;
      hipEventElapsedTime(&warmelapsedTime,warstart,warmstop);
      float nowwarmup=(warmelapsedTime);
      // nowwarmup = max()
      int nowiter=(350+nowwarmup-1)/nowwarmup;

      for(int out=0; out<nowiter; out++)
      {
        for(int i=0; i<l_warmupiteration; i++)
        {
          // execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
                // (__var_2__, width_y, width_x , __var_1__);
          execute_kernel<<<executeGridDim, executeBlockDim, executeSM>>>
            (__var_2__, __var_1__,  height, width_y, width_x);
          REAL* tmp = __var_2__;
          __var_2__=__var_1__;
          __var_1__= tmp;
        }       
      }
  }
  #endif 
  
  #ifdef PERSISTENTLAUNCH
  {
      // double accumulate=0;
      hipEventRecord(warstart,0);
      hipLaunchCooperativeKernel((void*)execute_kernel, executeGridDim, executeBlockDim, KernelArgsNULL, executeSM,0);
      hipEventRecord(warmstop,0);
      hipEventSynchronize(warmstop);
      cudaCheckError();
      float warmelapsedTime;
      hipEventElapsedTime(&warmelapsedTime,warstart,warmstop);
      int nowwarmup=warmelapsedTime;
      int nowiter=(350+nowwarmup-1)/nowwarmup;
      for(int i=0; i<nowiter; i++)
      {
        hipLaunchCooperativeKernel((void*)execute_kernel, executeGridDim, executeBlockDim, KernelArgsNULL, executeSM,0);
      }
  }
  #endif
}

#ifdef _TIMER_
  hipEvent_t _forma_timer_start_,_forma_timer_stop_;
  hipEventCreate(&_forma_timer_start_);
  hipEventCreate(&_forma_timer_stop_);
  hipEventRecord(_forma_timer_start_,0);
#endif


#ifdef TRADITIONLAUNCH
  execute_kernel<<<executeGridDim, executeBlockDim,executeSM>>>
          (input, __var_2__,  height, width_y, width_x);

  for(int i=1; i<iteration; i++)
  {
     execute_kernel<<<executeGridDim, executeBlockDim,executeSM>>>
          (__var_2__, __var_1__, height, width_y, width_x);
    REAL* tmp = __var_2__;
    __var_2__=__var_1__;
    __var_1__= tmp;
  }
#endif
#ifdef PERSISTENTLAUNCH
  hipLaunchCooperativeKernel((void*)execute_kernel, executeGridDim, executeBlockDim, KernelArgs, executeSM,0);
#endif
  hipDeviceSynchronize();
  cudaCheckError();
#ifdef _TIMER_
  hipEventRecord(_forma_timer_stop_,0);
  hipEventSynchronize(_forma_timer_stop_);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime,_forma_timer_start_,_forma_timer_stop_);
#ifndef __PRINT__
  printf("[FORMA] SIZE : %d,%d,%d\n",height,width_y,width_x);
  printf("[FORMA] Computation Time(ms) : %lf\n",elapsedTime);
  printf("[FORMA] Speed(GCells/s) : %lf\n",(REAL)iteration*height*width_x*width_y/ elapsedTime/1000/1000);
  printf("[FORMA] Computation(GFLOPS/s) : %lf\n",(REAL)iteration*height*width_x*width_y*(HALO*2+1)*(HALO*2+1)/ elapsedTime/1000/1000);
  printf("[FORMA] Bandwidht(GB/s) : %lf\n",(REAL)iteration*height*width_x*width_y*sizeof(REAL)*2/ elapsedTime/1000/1000);
  printf("[FORMA] rfder : %d\n",REG_FOLDER_Z);
#ifdef PERSISTENTLAUNCH
  printf("[FORMA] sfder : %d\n",max_sm_flder);
  // printf("[FORMA] sm : %f\n",executeSM/1024);
#endif
#else
  // h y x iter TILEX thready=1 gridx gridy latency speed 
  printf("%d\t%d\t",ptx,sizeof(REAL)/4);
  printf("%d\t%d\t%d\t%d\t",height,width_y,width_x,iteration); 
  printf("%d\t<%d,%d,%d>\t%d\t%d\t",executeBlockDim.x,
        executeGridDim.x,executeGridDim.y,executeGridDim.z,sm_count,
        (executeGridDim.x)*(executeGridDim.y)*(executeGridDim.z)/sm_count);
  #ifndef NAIVE
  printf("%f\t",(double)basic_sm_space/1024);
  #endif
  printf("%f\t%lf\n",elapsedTime,(REAL)iteration*height*width_x*width_y/ elapsedTime/1000/1000); 

#endif
  hipEventDestroy(_forma_timer_start_);
  hipEventDestroy(_forma_timer_stop_);
#endif
  hipDeviceSynchronize();
  cudaCheckError();
  
#if defined(PERSISTENTLAUNCH) 
// || defined(PERSISTENT)
  if(iteration%2==1)  
  {
    hipMemcpy(__var_0__, __var_2__, sizeof(REAL)*height*width_x*width_y, hipMemcpyDeviceToHost);
  }
  else
  {
    hipMemcpy(__var_0__, input, sizeof(REAL)*height*width_x*width_y, hipMemcpyDeviceToHost);
  }
#else
  hipMemcpy(__var_0__, __var_2__, sizeof(REAL)*height*width_x*width_y, hipMemcpyDeviceToHost);
#endif
  hipDeviceSynchronize();
  cudaCheckError();

  hipFree(input);
  hipFree(__var_1__);
  hipFree(__var_2__);
  hipFree(l2_cache1);
  hipFree(l2_cache2);
  return 0;
}

PERKS_INITIALIZE_ALL_TYPE(PERKS_DECLARE_INITIONIZATION_ITERATIVE);

// template void j3d_iterative<float>(float * h_input, int height, int width_y, int width_x, float * __var_0__, int iteration);
// template void j3d_iterative<double>(float * h_input, int height, int width_y, int width_x, float * __var_0__, int iteration);
