#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include "util/cub_utils.cuh"
// #include <cub_utils.cuh>
#include "cg_driver.cuh"

// #include "util/command.cuh"

int main(int argc, char **argv) {
  CommandLineArgs args(argc, argv);

  bool                fp32=true;
  int devID = findCudaDevice(argc, (const char **)argv);
  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
  fp32 = args.CheckCmdLineFlag("fp32");
  // printf("%d",fp32);
  if (fp32)
  {
    #ifndef __PRINT__
      printf("float\n");
      fprintf(stderr,"float\t");
    #endif
    myTest<float,int>(devID,deviceProp,args);
  }
  else
  {
    #ifndef __PRINT__
      printf("double\n");
      fprintf(stderr,"double\t");
    #endif
    myTest<double,int>(devID,deviceProp,args);
  }
  // exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}


