#include "hip/hip_runtime.h"
#include "cub_utils.cuh"

#include <string>
#include <cub/util_device.cuh>
#include <sstream>

namespace mersenne {

/* Period parameters */
const unsigned int N          = 624;
const unsigned int M          = 397;
const unsigned int MATRIX_A   = 0x9908b0df; /* constant vector a */
const unsigned int UPPER_MASK = 0x80000000; /* most significant w-r bits */
const unsigned int LOWER_MASK = 0x7fffffff; /* least significant r bits */

static unsigned int mt[N];  /* the array for the state vector  */
static int mti = N + 1;     /* mti==N+1 means mt[N] is not initialized */

/* initializes mt[N] with a seed */
void init_genrand(unsigned int s)
{
    mt[0] = s & 0xffffffff;
    for (mti = 1; mti < N; mti++)
    {
        mt[mti] = (1812433253 * (mt[mti - 1] ^ (mt[mti - 1] >> 30)) + mti);

        /* See Knuth TAOCP Vol2. 3rd Ed. P.106 for mtiplier. */
        /* In the previous versions, MSBs of the seed affect   */
        /* only MSBs of the array mt[].                        */
        /* 2002/01/09 modified by Makoto Matsumoto             */

        mt[mti] &= 0xffffffff;
        /* for >32 bit machines */
    }
}

/* initialize by an array with array-length */
/* init_key is the array for initializing keys */
/* key_length is its length */
/* slight change for C++, 2004/2/26 */
void init_by_array(unsigned int init_key[], int key_length)
{
    int i, j, k;
    init_genrand(19650218);
    i = 1;
    j = 0;
    k = (N > key_length ? N : key_length);
    for (; k; k--)
    {
        mt[i] = (mt[i] ^ ((mt[i - 1] ^ (mt[i - 1] >> 30)) * 1664525))
            + init_key[j] + j;  /* non linear */
        mt[i] &= 0xffffffff;    /* for WORDSIZE > 32 machines */
        i++;
        j++;
        if (i >= N)
        {
            mt[0] = mt[N - 1];
            i = 1;
        }
        if (j >= key_length) j = 0;
    }
    for (k = N - 1; k; k--)
    {
        mt[i] = (mt[i] ^ ((mt[i - 1] ^ (mt[i - 1] >> 30)) * 1566083941)) - i; /* non linear */
        mt[i] &= 0xffffffff; /* for WORDSIZE > 32 machines */
        i++;
        if (i >= N)
        {
            mt[0] = mt[N - 1];
            i = 1;
        }
    }

    mt[0] = 0x80000000; /* MSB is 1; assuring non-zero initial array */
}

/* generates a random number on [0,0xffffffff]-interval */
unsigned int genrand_int32(void)
{
    unsigned int y;
    static unsigned int mag01[2] = { 0x0, MATRIX_A };

    /* mag01[x] = x * MATRIX_A  for x=0,1 */

    if (mti >= N)
    { /* generate N words at one time */
        int kk;

        if (mti == N + 1) /* if init_genrand() has not been called, */
        init_genrand(5489); /* a defat initial seed is used */

        for (kk = 0; kk < N - M; kk++)
        {
            y = (mt[kk] & UPPER_MASK) | (mt[kk + 1] & LOWER_MASK);
            mt[kk] = mt[kk + M] ^ (y >> 1) ^ mag01[y & 0x1];
        }
        for (; kk < N - 1; kk++)
        {
            y = (mt[kk] & UPPER_MASK) | (mt[kk + 1] & LOWER_MASK);
            mt[kk] = mt[kk + (M - N)] ^ (y >> 1) ^ mag01[y & 0x1];
        }
        y = (mt[N - 1] & UPPER_MASK) | (mt[0] & LOWER_MASK);
        mt[N - 1] = mt[M - 1] ^ (y >> 1) ^ mag01[y & 0x1];

        mti = 0;
    }

    y = mt[mti++];

    /* Tempering */
    y ^= (y >> 11);
    y ^= (y << 7) & 0x9d2c5680;
    y ^= (y << 15) & 0xefc60000;
    y ^= (y >> 18);

    return y;
}



} // namespace mersenne


// struct CommandLineArgs
// {

//     std::vector<std::string>    keys;
//     std::vector<std::string>    values;
//     std::vector<std::string>    args;
// #ifdef __NVCC__
//     hipDeviceProp_t              deviceProp;
// #endif // __NVCC__
//     float                       device_giga_bandwidth;
//     size_t                      device_free_physmem;
//     size_t                      device_total_physmem;

    /**
     * Constructor
     */
    CommandLineArgs::CommandLineArgs(int argc, char **argv)// :
        // keys(10),
        // values(10)
    {
        // this->keys(10);
        // this->values(10);
        using namespace std;

        // Initialize mersenne generator
        unsigned int mersenne_init[4]=  {0x123, 0x234, 0x345, 0x456};
        mersenne::init_by_array(mersenne_init, 4);

        for (int i = 1; i < argc; i++)
        {
            string arg = argv[i];

            if ((arg[0] != '-') || (arg[1] != '-'))
            {
                args.push_back(arg);
                continue;
            }

            string::size_type pos;
            string key, val;
            if ((pos = arg.find('=')) == string::npos) {
                key = string(arg, 2, arg.length() - 2);
                val = "";
            } else {
                key = string(arg, 2, pos - 2);
                val = string(arg, pos + 1, arg.length() - 1);
            }

            keys.push_back(key);
            values.push_back(val);
        }
    }


    /**
     * Checks whether a flag "--<flag>" is present in the commandline
     */
    bool CommandLineArgs::CheckCmdLineFlag(const char* arg_name)
    {
        using namespace std;

        for (int i = 0; i < int(keys.size()); ++i)
        {
            if (keys[i] == string(arg_name))
                return true;
        }
        return false;
    }


    /**
     * Returns number of naked (non-flag and non-key-value) commandline parameters
     */
    template <typename T>
    int CommandLineArgs::NumNakedArgs()
    {
        return args.size();
    }


    /**
     * Returns the commandline parameter for a given index (not including flags)
     */
    template <typename T>
    void CommandLineArgs::GetCmdLineArgument(int index, T &val)
    {
        using namespace std;
        if (index < args.size()) {
            istringstream str_stream(args[index]);
            str_stream >> val;
        }
    }

    /**
     * Returns the value specified for a given commandline parameter --<flag>=<value>
     */
    template <typename T>
    void CommandLineArgs::GetCmdLineArgument(const char *arg_name, T &val)
    {
        using namespace std;

        for (int i = 0; i < int(keys.size()); ++i)
        {
            if (keys[i] == string(arg_name))
            {
                istringstream str_stream(values[i]);
                str_stream >> val;
            }
        }
    }


    template void CommandLineArgs::GetCmdLineArgument<std::string>(const char *arg_name, std::string &val);
    /**
     * Returns the values specified for a given commandline parameter --<flag>=<value>,<value>*
     */
    template <typename T>
    void CommandLineArgs::GetCmdLineArguments(const char *arg_name, std::vector<T> &vals)
    {
        using namespace std;

        if (CheckCmdLineFlag(arg_name))
        {
            // Clear any default values
            vals.clear();

            // Recover from multi-value string
            for (int i = 0; i < keys.size(); ++i)
            {
                if (keys[i] == string(arg_name))
                {
                    string val_string(values[i]);
                    istringstream str_stream(val_string);
                    string::size_type old_pos = 0;
                    string::size_type new_pos = 0;

                    // Iterate comma-separated values
                    T val;
                    while ((new_pos = val_string.find(',', old_pos)) != string::npos)
                    {
                        if (new_pos != old_pos)
                        {
                            str_stream.width(new_pos - old_pos);
                            str_stream >> val;
                            vals.push_back(val);
                        }

                        // skip over comma
                        str_stream.ignore(1);
                        old_pos = new_pos + 1;
                    }

                    // Read last value
                    str_stream >> val;
                    vals.push_back(val);
                }
            }
        }
    }

    // template void CommandLineArgs::GetCmdLineArgument<char>(const char *arg_name, std::vector<char> &val);

    /**
     * The number of pairs parsed
     */
    int CommandLineArgs::ParsedArgc()
    {
        return (int) keys.size();
    }


/**
 * Compares the equivalence of two arrays
 */
template <typename S, typename T, typename OffsetT>
int CompareResults(T* computed, S* reference, OffsetT len, bool verbose = true)
{
    for (OffsetT i = 0; i < len; i++)
    {
        if (computed[i] != reference[i])
        {
            if (verbose) std::cout << "INCORRECT: [" << i << "]: "
                << computed[i] << " != "
                << reference[i];
            return 1;
        }
    }
    return 0;
}


/**
 * Compares the equivalence of two arrays
 */
template <typename OffsetT>
int CompareResults(float* computed, float* reference, OffsetT len, bool verbose = true)
{
    // float meps = std::numeric_limits<float>::epsilon();
 
    for (OffsetT i = 0; i < len; i++)
    {
        float   a           = computed[i];
        float   b           = reference[i];
        int     int_diff    = std::abs(*(int*)&a - *(int*)&b);
        float   sqrt_diff   = sqrt(float(int_diff));
        if(i<20)
        {
            printf("<<<%f,%f>>>\n",a,b);
        }
        if (sqrt_diff > len)      
        {
            if (verbose) std::cout << "INCORRECT (sqrt_diff: " << sqrt_diff << "): [" << i << "]: "
                 << computed[i] << " != "
                 << reference[i]; 
            return 1;
        }
    }
    return 0;
}



/**
 * Compares the equivalence of two arrays
 */
template <typename OffsetT>
int CompareResults(double* computed, double* reference, OffsetT len, bool verbose = true)
{
    // double meps = std::numeric_limits<double>::epsilon();
    // float fmeps = std::numeric_limits<float>::epsilon();
 
    for (OffsetT i = 0; i < len; i++)
    {
        float   a           = computed[i];
        float   b           = reference[i];
        int     int_diff    = std::abs(*(int*)&a - *(int*)&b);
        float   sqrt_diff   = sqrt(float(int_diff));
        if(i<20)
        {
            printf("<<<%f,%f,%d,%f>>>\n",a,b,int_diff,sqrt_diff);
        }
        if (sqrt_diff > len)      
        {
            if (verbose) std::cout << "INCORRECT (sqrt_diff: " << sqrt_diff << "): [" << i << "]: "
                 << computed[i] << " != "
                 << reference[i]; 
            return 1;
        }
    }
    return 0;
}



#ifdef __NVCC__

    /**
     * Initialize device
     */
    hipError_t CommandLineArgs::DeviceInit(int dev)
    {
        hipError_t error = hipSuccess;

        do
        {
            int deviceCount;
            error = HipcubDebug(hipGetDeviceCount(&deviceCount));
            if (error) break;

            if (deviceCount == 0) {
                fprintf(stderr, "No devices supporting CUDA.\n");
                exit(1);
            }
            if (dev < 0)
            {
                GetCmdLineArgument("device", dev);
            }
            if ((dev > deviceCount - 1) || (dev < 0))
            {
                dev = 0;
            }

            error = HipcubDebug(hipSetDevice(dev));
            if (error) break;

            CubDebugExit(hipMemGetInfo(&device_free_physmem, &device_total_physmem));

            int ptx_version;
            error = HipcubDebug(hipcub::PtxVersion(ptx_version));
            if (error) break;

            error = HipcubDebug(hipGetDeviceProperties(&deviceProp, dev));
            if (error) break;

            if (deviceProp.major < 1) {
                fprintf(stderr, "Device does not support CUDA.\n");
                exit(1);
            }

            device_giga_bandwidth = float(deviceProp.memoryBusWidth) * deviceProp.memoryClockRate * 2 / 8 / 1000 / 1000;

            if (!CheckCmdLineFlag("quiet"))
            {
                printf(
                        "Using device %d: %s (PTX version %d, SM%d, %d SMs, "
                        "%lld free / %lld total MB physmem, "
                        "%.3f GB/s @ %d kHz mem clock, ECC %s)\n",
                    dev,
                    deviceProp.name,
                    ptx_version,
                    deviceProp.major * 100 + deviceProp.minor * 10,
                    deviceProp.multiProcessorCount,
                    (unsigned long long) device_free_physmem / 1024 / 1024,
                    (unsigned long long) device_total_physmem / 1024 / 1024,
                    device_giga_bandwidth,
                    deviceProp.memoryClockRate,
                    (deviceProp.ECCEnabled) ? "on" : "off");
                fflush(stdout);
            }

        } while (0);

        return error;
    }


template <typename T>
void DisplayDeviceResults(
    T *d_data,
    size_t num_items)
{
    // Allocate array on host
    T *h_data = (T*) malloc(num_items * sizeof(T));

    // Copy data back
    hipMemcpy(h_data, d_data, sizeof(T) * num_items, hipMemcpyDeviceToHost);

    DisplayResults(h_data, num_items);

    // Cleanup
    if (h_data) free(h_data);
}

/**
 * Verify the contents of a device array match those
 * of a host array
 */
template <typename S, typename T>
int CompareDeviceResults(
    S *h_reference,
    T *d_data,
    size_t num_items,
    bool verbose ,
    bool display_data )
{
    // Allocate array on host
    T *h_data = (T*) malloc(num_items * sizeof(T));

    // Copy data back
    hipMemcpy(h_data, d_data, sizeof(T) * num_items, hipMemcpyDeviceToHost);

    // Display data
    if (display_data)
    {
        printf("Reference:\n");
        for (int i = 0; i < int(num_items); i++)
        {
            std::cout << h_reference[i] << ", ";
        }
        printf("\n\nComputed:\n");
        for (int i = 0; i < int(num_items); i++)
        {
            std::cout << h_data[i] << ", ";
        }
        printf("\n\n");
    }

    // Check
    int retval = CompareResults(h_data, h_reference, num_items, verbose);

    // Cleanup
    if (h_data) free(h_data);

    return retval;
}


/**
 * Verify the contents of a device array match those
 * of a device array
 */
template <typename T>
int CompareDeviceDeviceResults(
    T *d_reference,
    T *d_data,
    size_t num_items,
    bool verbose ,
    bool display_data )
{
    // Allocate array on host
    T *h_reference = (T*) malloc(num_items * sizeof(T));
    T *h_data = (T*) malloc(num_items * sizeof(T));

    // Copy data back
    hipMemcpy(h_reference, d_reference, sizeof(T) * num_items, hipMemcpyDeviceToHost);
    hipMemcpy(h_data, d_data, sizeof(T) * num_items, hipMemcpyDeviceToHost);

    // Display data
    if (display_data) {
        printf("Reference:\n");
        for (int i = 0; i < num_items; i++)
        {
            std::cout << (h_reference[i]) << ", ";
            //std::cout << CoutCast(h_reference[i]) << ", ";
        }
        printf("\n\nComputed:\n");
        for (int i = 0; i < num_items; i++)
        {
            //std::cout << CoutCast(h_data[i]) << ", ";
            std::cout << (h_data[i]) << ", ";
        }
        printf("\n\n");
    }

    // Check
    int retval = CompareResults(h_data, h_reference, num_items, verbose);

    // Cleanup
    if (h_reference) free(h_reference);
    if (h_data) free(h_data);

    return retval;
}

template<class TYPE>
bool areAlmostEqual(TYPE a, TYPE b, TYPE maxRelDiff) {
  TYPE diff = fabsf(a - b);
  TYPE abs_a = fabsf(a);
  TYPE abs_b = fabsf(b);
  TYPE largest = abs_a > abs_b ? abs_a : abs_b;

  if (diff <= largest * maxRelDiff) {
    return true;
  } else {
    printf("maxRelDiff = %.8e\n", maxRelDiff);
    printf(
        "diff %.8e > largest * maxRelDiff %.8e therefore %.8e and %.8e are not "
        "same\n",
        diff, largest * maxRelDiff, a, b);
    return false;
  }
}


// template void DisplayDeviceResults<float>(float *d_data,size_t num_items);
// template void DisplayDeviceResults<double>(double *d_data,size_t num_items);

template int CompareDeviceResults<float,float>(
    float *h_reference,
    float *d_data,
    size_t num_items,
    bool verbose,
    bool display_data);
template int CompareDeviceResults<double,double>(
    double *h_reference,
    double *d_data,
    size_t num_items,
    bool verbose,
    bool display_data);

template int CompareDeviceDeviceResults<float>(
    float *d_reference,
    float *d_data,
    size_t num_items,
    bool verbose ,
    bool display_data );
template int CompareDeviceDeviceResults<double>(
    double *d_reference,
    double *d_data,
    size_t num_items,
    bool verbose ,
    bool display_data );

template bool areAlmostEqual<float>(float a, float b, float maxRelDiff);
template bool areAlmostEqual<double>(double a, double b, double maxRelDiff);

#endif // __NVCC__

// };
